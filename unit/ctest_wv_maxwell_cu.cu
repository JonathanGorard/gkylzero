#include "hip/hip_runtime.h"
/* -*- c -*- */

#include <stdio.h>
#include <gkylzero.h>
#include <gkyl_wv_maxwell.h>
#include <gkyl_wv_maxwell_priv.h>

// Make indexing cleaner with the dir_shuffle
#define EX 0
#define EY 1
#define EZ 2
#define BX 3
#define BY 4
#define BZ 5

extern "C" {
    int cu_maxwell_test(const struct gkyl_wv_eqn *eqn);
}

__global__
void ker_cu_maxwell_test(const struct gkyl_wv_eqn *eqn, int *nfail)
{
  *nfail = 0;

  GKYL_CU_CHECK( eqn->num_equations == 8, nfail );
  GKYL_CU_CHECK( eqn->num_waves == 6 );

  // DO NOT DO THIS IN PRODUCTION! ONLY FOR TESTING
  struct wv_maxwell *maxwell = container_of(eqn, struct wv_maxwell, eqn);

  GKYL_CU_CHECK( maxwell->c == 299792458.0, nfail );
  GKYL_CU_CHECK( maxwell->e_fact == 2.0, nfail );
  GKYL_CU_CHECK( maxwell->b_fact == 2.5, nfail );

  double Ex = 1.0, Ey = 0.1, Ez = 0.2;
  double Bx = 10.0, By = 10.1, Bz = 10.2;
  double phi = 0.01, psi = 0.02;
  double q[8] = { Ex, Ey, Ez, Bx, By, Bz, phi, psi };

  double norm[3][3] = {
    { 1.0, 0.0, 0.0 },
    { 0.0, 1.0, 0.0 },
    { 0.0, 0.0, 1.0 }
  };

  double tau1[3][3] = {
    { 0.0, 1.0, 0.0 },
    { 1.0, 0.0, 0.0 },
    { 1.0, 0.0, 0.0 }
  };

  double tau2[3][3] = {
    { 0.0, 0.0, 1.0 },
    { 0.0, 0.0, -1.0 },
    { 0.0, 1.0, 0.0 }
  };

  double fluxes[3][8] = {
    {
      e_fact*c2*q[6],
      c2*q[BZ],
      -c2*q[BY],
      
      b_fact*q[7],
      -q[EZ],
      q[EY],
      
      e_fact*q[EX],
      b_fact*c2*q[BX]
    },
    {
      -c2*q[BZ],
      e_fact*c2*q[6],
      c2*q[BX],
      
      q[EZ],
      b_fact*q[7],
      -q[EX],
      
      e_fact*q[EY],
      b_fact*c2*q[BY]
    },
    {
      c2*q[BY],
      -c2*q[BX],      
      e_fact*c2*q[6],
      
      -q[EY],
      q[EX],
      b_fact*q[7],
      
      e_fact*q[EZ],
      b_fact*c2*q[BZ]
    },

  };


  double q_local[8], flux_local[8], flux[8];

  for (int d=0; d<3; ++d) {
    eqn->rotate_to_local_func(tau1[d], tau2[d], norm[d], q, q_local);
    gkyl_maxwell_flux(c, e_fact, b_fact, q_local, flux_local);
    eqn->rotate_to_global_func(tau1[d], tau2[d], norm[d], flux_local, flux);

    for (int m=0; m<8; ++m)
      GKYL_CU_CHECK( gkyl_compare(flux[m], fluxes[d][m], 1e-15) , nfail );
  }
}

int cu_maxwell_test(const struct gkyl_wv_eqn *eqn)
{
  int *nfail_dev = (int *) gkyl_cu_malloc(sizeof(int));
  ker_cu_maxwell_test<<<1,1>>>(eqn, nfail_dev);

  int nfail;
  gkyl_cu_memcpy(&nfail, nfail_dev, sizeof(int), GKYL_CU_MEMCPY_D2H);
  gkyl_cu_free(nfail_dev);

  return nfail;
}

