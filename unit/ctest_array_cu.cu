#include "hip/hip_runtime.h"
/* -*- c -*- */

#include <gkylzero.h>
#include <stdio.h>

extern "C" {
    int cu_array_test_and_flip_sign( struct gkyl_array *arr);
}

__global__
void ker_cu_array_test_and_flip_sign( struct gkyl_array *arr, int *nfail)
{
  *nfail = 0;
  
  GKYL_CU_CHECK( arr->type == GKYL_DOUBLE, nfail );
  GKYL_CU_CHECK( arr->elemsz ==sizeof(double), nfail );
  GKYL_CU_CHECK( arr->ncomp == 1, nfail );
  GKYL_CU_CHECK( arr->size == 20, nfail );
  
   double *data = ( double *) arr->data;
  for (unsigned i=0; i<arr->size; ++i) {
    GKYL_CU_CHECK( data[i] == (i+0.5)*0.1, nfail );
    data[i] *= -1;
  }

}

int cu_array_test_and_flip_sign( struct gkyl_array *arr)
{
  int *nfail_dev = (int *) gkyl_cu_malloc(sizeof(int));
  ker_cu_array_test_and_flip_sign<<<1,1>>>(arr, nfail_dev);

  int nfail;
  gkyl_cu_memcpy(&nfail, nfail_dev, sizeof(int), GKYL_CU_MEMCPY_D2H);
  gkyl_cu_free(nfail_dev);

  return nfail;
}
