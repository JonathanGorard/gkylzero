#include "hip/hip_runtime.h"
#include <gkylzero.h>
#include <gkyl_dg_vlasov_priv.h>

#define TEST_NO_MAIN
#include <acutest.h>

extern "C" {
void test_vlasov_2x3v_p1_cu();
}

__global__
void ker_cu_hyper_dg_test(const gkyl_hyper_dg *slvr, int *nfail)
{
  *nfail = 0;
  
  GKYL_CU_CHECK( slvr->ndim == 5, nfail );
  GKYL_CU_CHECK( slvr->numBasis == 32, nfail );
  GKYL_CU_CHECK( slvr->num_up_dirs == 5, nfail );

  GKYL_CU_CHECK( slvr->grid.ndim == 5, nfail );

  GKYL_CU_CHECK( slvr->equation->num_equations == 1, nfail );

  // DO NOT DO THIS IN PRODUCTION! ONLY FOR TESTING
  struct dg_vlasov *vlasov = container_of(slvr->equation, struct dg_vlasov, eqn);

  GKYL_CU_CHECK( vlasov->cdim == 2, nfail );
  GKYL_CU_CHECK( vlasov->pdim == 5, nfail );
  GKYL_CU_CHECK( vlasov->conf_range.volume == 8*8, nfail );
}

static int
hyper_dg_test(const gkyl_hyper_dg *slvr)
{
  int *nfail_dev = (int *) gkyl_cu_malloc(sizeof(int));
  ker_cu_hyper_dg_test<<<1,1>>>(slvr, nfail_dev);

  int nfail;
  gkyl_cu_memcpy(&nfail, nfail_dev, sizeof(int), GKYL_CU_MEMCPY_D2H);
  gkyl_cu_free(nfail_dev);

  return nfail;  
}

// allocate array (filled with zeros)
static struct gkyl_array*
mkarr(long nc, long size)
{
  struct gkyl_array* a = gkyl_array_new(GKYL_DOUBLE, nc, size);
  return a;
}

// allocate cu_dev array
static struct gkyl_array*
mkarr_cu(long nc, long size)
{
  struct gkyl_array* a = gkyl_array_cu_dev_new(GKYL_DOUBLE, nc, size);
  return a;
}

void test_vlasov_2x3v_p1_cu()
{
  // initialize grid and ranges on host
  int cdim = 2, vdim = 3;
  int pdim = cdim+vdim;

  int cells[] = {8, 8, 8, 8, 8};
  int ghost[] = {1, 1, 0, 0, 0};
  double lower[] = {0., 0., -1., -1., -1.};
  double upper[] = {1., 1., 1., 1., 1.};

  struct gkyl_rect_grid confGrid;
  struct gkyl_range confRange, confRange_ext;
  gkyl_rect_grid_init(&confGrid, cdim, lower, upper, cells);
  gkyl_create_grid_ranges(&confGrid, ghost, &confRange_ext, &confRange);

  struct gkyl_rect_grid phaseGrid;
  struct gkyl_range phaseRange, phaseRange_ext;
  gkyl_rect_grid_init(&phaseGrid, pdim, lower, upper, cells);
  gkyl_create_grid_ranges(&phaseGrid, ghost, &phaseRange_ext, &phaseRange);

  // clone grid and ranges to device
  struct gkyl_rect_grid *confGrid_cu = gkyl_rect_grid_clone_on_cu_dev(&confGrid);    
  struct gkyl_rect_grid *phaseGrid_cu = gkyl_rect_grid_clone_on_cu_dev(&phaseGrid);    
  struct gkyl_range *confRange_cu = gkyl_range_clone_on_cu_dev(&confRange);
  struct gkyl_range *confRange_ext_cu = gkyl_range_clone_on_cu_dev(&confRange_ext);
  struct gkyl_range *phaseRange_cu = gkyl_range_clone_on_cu_dev(&phaseRange);
  struct gkyl_range *phaseRange_ext_cu = gkyl_range_clone_on_cu_dev(&phaseRange_ext);

  // initialize basis (note: basis has no device implementation)
  int poly_order = 1;
  struct gkyl_basis basis, confBasis; // phase-space, conf-space basis

  gkyl_cart_modal_serendip(&basis, pdim, poly_order);
  gkyl_cart_modal_serendip(&confBasis, cdim, poly_order);

  // initialize eqn on device
  struct gkyl_dg_eqn *eqn_cu;
  eqn_cu = gkyl_dg_vlasov_cu_dev_new(&confBasis, &basis, confRange_cu);

  // initialize hyper_dg slvr
  int up_dirs[] = {0, 1, 2, 3, 4};
  int zero_flux_flags[] = {0, 0, 1, 1, 1};

  gkyl_hyper_dg *slvr_cu;
  slvr_cu = gkyl_hyper_dg_cu_dev_new(phaseGrid_cu, &basis, eqn_cu, pdim, up_dirs, zero_flux_flags, 1);

  // basic checks
  int nfail = hyper_dg_test(slvr_cu);

  TEST_CHECK( nfail == 0 );

  // initialize host arrays
  struct gkyl_array *fin, *rhs, *cflrate, *qmem;
  fin = mkarr(basis.numBasis, phaseRange_ext.volume);
  rhs = mkarr(basis.numBasis, phaseRange_ext.volume);
  cflrate = mkarr(1, phaseRange_ext.volume);
  qmem = mkarr(8*confBasis.numBasis, confRange_ext.volume);

  // set initial condition
  int nf = phaseRange_ext.volume*basis.numBasis;
  double *fin_d = (double*) fin->data;
  for(int i=0; i< nf; i++) {
    fin_d[i] = (double)(2*i+11 % nf) / nf  * ((i%2 == 0) ? 1 : -1);
  }

  int nem = confRange_ext.volume*confBasis.numBasis;
  double *qmem_d = (double*) qmem->data;
  for(int i=0; i< nem; i++) {
    qmem_d[i] = (double)(-i+27 % nem) / nem  * ((i%2 == 0) ? 1 : -1);
  }

  // initialize device arrays 
  struct gkyl_array *fin_cu = mkarr_cu(basis.numBasis, phaseRange_ext.volume);
  struct gkyl_array *rhs_cu = mkarr_cu(basis.numBasis, phaseRange_ext.volume);
  struct gkyl_array *cflrate_cu = mkarr_cu(1, phaseRange_ext.volume);
  struct gkyl_array *qmem_cu = mkarr_cu(8*confBasis.numBasis, confRange_ext.volume);

  // copy initial conditions to device
  gkyl_array_copy(fin_cu, fin);
  gkyl_array_copy(qmem_cu, qmem);

  // maxs_cu is not an array struct, just a regular array
  double *maxs_cu = (double*) gkyl_cu_malloc(sizeof(double)*5);

  // run hyper_dg_advance
  int nrep = 10;
  for(int n=0; n<nrep; n++) {
    // zero out array struct device data (eventually these will be handled by array_ops)
    hipMemset(rhs_cu->data, 0.0, rhs_cu->size*rhs_cu->esznc);
    hipMemset(cflrate_cu->data, 0.0, cflrate_cu->size*cflrate_cu->esznc);

    // also zero out maxs_cu
    hipMemset(maxs_cu, 0., sizeof(double)*5);

    // set pointer to EM fields in vlasov equation object (on device)
    gkyl_vlasov_set_qmem_cu(eqn_cu, qmem_cu->on_device); // must set EM fields to use

    int dB = 256;
    int dG = phaseRange.volume/dB + 1;
    gkyl_hyper_dg_advance_cu<<<dG,dB>>>(slvr_cu, phaseRange_cu, fin_cu->on_device, cflrate_cu->on_device, rhs_cu->on_device, maxs_cu);
  }

  // copy result from device to host
  gkyl_array_copy(rhs, rhs_cu);

  // get linear index of first non-ghost cell
  int idx[] = {0, 0, 0, 0, 0};
  int linl = gkyl_range_idx(&phaseRange, idx);

  // check that ghost cells are empty
  double val = 0;
  double *rhs_d;
  int i = 0;
  while(val==0) {
    rhs_d = (double*) gkyl_array_fetch(rhs, i);
    val = rhs_d[1];
    if(val==0) i++;
  }
  TEST_CHECK(i == linl);

  // check data in first non-ghost cell
  rhs_d = (double*) gkyl_array_fetch(rhs, linl);
  TEST_CHECK( gkyl_compare_double(rhs_d[0],  4.894191610931403e+00 , 1e-12) ); 
  TEST_CHECK( gkyl_compare_double(rhs_d[1],  1.331341236610166e+01 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[2],  -8.324741199843084e+00, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[3],  -2.673619244471518e+00, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[4],  4.243853346589546e+00 , 1e-12) ); 
  TEST_CHECK( gkyl_compare_double(rhs_d[5],  -6.618097617712298e+00, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[6],  -2.204812073157034e+01, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[7],  1.233289696343498e+01 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[8],  -1.462768984400967e+01, 1e-12) ); 
  TEST_CHECK( gkyl_compare_double(rhs_d[9],  1.711468109826621e+01 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[10], -4.311578661561084e+00, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[11], -1.055566810350123e+01, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[12], -1.462989088297130e+01, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[13], 3.475894627821628e+00 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[14], 1.049594917536618e+01 , 1e-12) ); 
  TEST_CHECK( gkyl_compare_double(rhs_d[15], 9.761290631219577e-01 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[16], -1.621943928500327e+01, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[17], 2.722448306561996e+01 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[18], -1.069595454463273e+01, 1e-12) ); 
  TEST_CHECK( gkyl_compare_double(rhs_d[19], -4.546812665639302e-01, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[20], -2.315401840681718e+01, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[21], 1.084202513678910e+01 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[22], 1.965414256580261e+00 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[23], 9.307169707410683e+00 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[24], -1.159443470189338e+01, 1e-12) ); 
  TEST_CHECK( gkyl_compare_double(rhs_d[25], 8.959151842026718e-01 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[26], -1.922720351907200e+01, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[27], 1.939005961012885e+01 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[28], -2.187959418345774e+01, 1e-12) ); 
  TEST_CHECK( gkyl_compare_double(rhs_d[29], 9.872685698935690e+00 , 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[30], -4.888428376000754e+00, 1e-12) );
  TEST_CHECK( gkyl_compare_double(rhs_d[31], 2.508827706127977e+01 , 1e-12) );

  // clean up
  gkyl_array_release(fin);
  gkyl_array_release(rhs);
  gkyl_array_release(cflrate);
  gkyl_array_release(qmem);

  gkyl_array_release(fin_cu);
  gkyl_array_release(rhs_cu);
  gkyl_array_release(cflrate_cu);
  gkyl_array_release(qmem_cu);

//  gkyl_hyper_dg_release(slvr_cu);
//  gkyl_dg_eqn_release(eqn_cu);
}
