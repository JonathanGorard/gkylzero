#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include "gkyl_alloc_flags_priv.h"
#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_array_ops.h>
#include <gkyl_hyper_dg.h>
#include <gkyl_hyper_dg_priv.h>
#include <gkyl_range.h>
#include <gkyl_rect_grid.h>
#include <gkyl_util.h>
}

__global__ static void
gkyl_hyper_dg_set_update_vol_cu_kernel(gkyl_hyper_dg *hdg, int update_vol_term)
{
  hdg->update_vol_term = update_vol_term;
}

__global__ static void
gkyl_hyper_dg_advance_cu_kernel(gkyl_hyper_dg* hdg, struct gkyl_range update_range,
  const struct gkyl_array* GKYL_RESTRICT fIn, struct gkyl_array* GKYL_RESTRICT cflrate,
  struct gkyl_array* GKYL_RESTRICT rhs)
{
  int ndim = hdg->ndim;
  int idxl[GKYL_MAX_DIM], idxc[GKYL_MAX_DIM], idxr[GKYL_MAX_DIM];
  double xcl[GKYL_MAX_DIM], xcc[GKYL_MAX_DIM], xcr[GKYL_MAX_DIM];
  // integer used for selecting between left-edge zero-flux BCs and right-edge zero-flux BCs
  int edge;

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < update_range.volume; linc1 += blockDim.x*gridDim.x) {
    // inverse index from linc1 to idxc
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idxc={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&update_range, linc1, idxc);
    gkyl_rect_grid_cell_center(&hdg->grid, idxc, xcc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long linc = gkyl_range_idx(&update_range, idxc);

    if (hdg->update_vol_term) {
      double cflr = hdg->equation->vol_term(
        hdg->equation, xcc, hdg->grid.dx, idxc,
        (const double*) gkyl_array_cfetch(fIn, linc), (double*) gkyl_array_fetch(rhs, linc)
      );
      double *cflrate_d = (double*) gkyl_array_fetch(cflrate, linc);
      cflrate_d[0] += cflr; // frequencies are additive
    }
    
    for (int d=0; d<hdg->num_up_dirs; ++d) {
      int dir = hdg->update_dirs[d];
      double cfls = 0.0;
      gkyl_copy_int_arr(ndim, idxc, idxl);
      gkyl_copy_int_arr(ndim, idxc, idxr);
      // TODO: fix for arbitrary subrange
      if (hdg->zero_flux_flags[dir] && (idxc[dir] == update_range.lower[dir] || idxc[dir] == update_range.upper[dir])) {
        edge = (idxc[dir] == update_range.lower[dir]) ? -1 : 1;
        // use idxl to store interior edge index (first index away from skin cell)
        idxl[dir] = idxl[dir]-edge;

        gkyl_rect_grid_cell_center(&hdg->grid, idxl, xcl);
        long linl = gkyl_range_idx(&update_range, idxl);

        cfls = hdg->equation->boundary_surf_term(hdg->equation,
          dir, xcl, xcc, hdg->grid.dx, hdg->grid.dx,
          idxl, idxc, edge,
          (const double*) gkyl_array_cfetch(fIn, linl), (const double*) gkyl_array_cfetch(fIn, linc),
          (double*) gkyl_array_fetch(rhs, linc)
        );
      }
      else {
        idxl[dir] = idxl[dir]-1;
        idxr[dir] = idxr[dir]+1;
        gkyl_rect_grid_cell_center(&hdg->grid, idxl, xcl);
        gkyl_rect_grid_cell_center(&hdg->grid, idxr, xcr);
        long linl = gkyl_range_idx(&update_range, idxl); 
        long linr = gkyl_range_idx(&update_range, idxr);

        cfls = hdg->equation->surf_term(hdg->equation,
          dir, xcl, xcc, xcr, hdg->grid.dx, hdg->grid.dx, hdg->grid.dx,
          idxl, idxc, idxr,
          (const double*) gkyl_array_cfetch(fIn, linl), (const double*) gkyl_array_cfetch(fIn, linc),
          (const double*) gkyl_array_cfetch(fIn, linr), (double*) gkyl_array_fetch(rhs, linc)
        );
      }
      double *cflrate_d = (double*) gkyl_array_fetch(cflrate, linc);
      cflrate_d[0] += cfls; // frequencies are additive     
    }
  }
}

// wrapper to call advance kernel on device
void
gkyl_hyper_dg_advance_cu(gkyl_hyper_dg* hdg, const struct gkyl_range *update_range,
  const struct gkyl_array* GKYL_RESTRICT fIn, struct gkyl_array* GKYL_RESTRICT cflrate,
  struct gkyl_array* GKYL_RESTRICT rhs)
{
  int nblocks = update_range->nblocks;
  int nthreads = update_range->nthreads;

  gkyl_hyper_dg_advance_cu_kernel<<<nblocks, nthreads>>>(hdg->on_dev, *update_range,
    fIn->on_dev, cflrate->on_dev, rhs->on_dev);
}

__global__ static void
gkyl_hyper_dg_gen_stencil_advance_cu_kernel(gkyl_hyper_dg* hdg, struct gkyl_range update_range,
  const struct gkyl_array* GKYL_RESTRICT fIn, struct gkyl_array* GKYL_RESTRICT cflrate,
  struct gkyl_array* GKYL_RESTRICT rhs)
{
  int ndim = hdg->ndim;

  // idxc, xc, and dx for volume update
  int idxc[GKYL_MAX_DIM] = {0};
  double xcc[GKYL_MAX_DIM] = {0.0};

  // idx for generic surface update
  int idx[9][GKYL_MAX_DIM] = {0};
  const double* fIn_d[9];

  // bool for checking if index is in the domain
  int in_grid = 1;

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < update_range.volume; linc1 += blockDim.x*gridDim.x) {
    // inverse index from linc1 to idxc
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idxc={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&update_range, linc1, idxc);
    gkyl_rect_grid_cell_center(&hdg->grid, idxc, xcc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long linc = gkyl_range_idx(&update_range, idxc);

    // Call volume kernel and get CFL rate
    gkyl_rect_grid_cell_center(&hdg->grid, idxc, xcc);
    double cflr = hdg->equation->vol_term(
      hdg->equation, xcc, hdg->grid.dx, idxc,
      (const double*) gkyl_array_cfetch(fIn, linc), (double*) gkyl_array_fetch(rhs, linc)
    );
    double *cflrate_d = gkyl_array_fetch(cflrate, linc);
    cflrate_d[0] += cflr;
    for (int d1=0; d1<hdg->num_up_dirs; ++d1) {
      for (int d2=0; d2<hdg->num_up_dirs; ++d2) {
        int dir1 = hdg->update_dirs[d1];
        int dir2 = hdg->update_dirs[d2];
        int update_dirs[2];
        update_dirs[0] = dir1;
        update_dirs[1] = dir2;

        long offsets[9] = {0};
        int keri = 0;

        // Create offsets for 2D stencil
        if (dir1 != dir2) {
          int num_up_dirs = 2;
          create_offsets(hdg, num_up_dirs, update_dirs, update_range, idxc, offsets);

          // Index into kernel list
          keri = idx_to_inloup_ker(num_up_dirs, idxc, update_dirs, update_range->upper);
        } 
        else {
          int num_up_dirs = 1;
          create_offsets(hdg, num_up_dirs, update_dirs, update_range, idxc, offsets);

          // Index into kernel list
          keri = idx_to_inloup_ker(num_up_dirs, idxc, update_dirs, update_range->upper);
        }

        // Get pointers to all neighbor values
        for (int i=0; i<9; ++i) {
          gkyl_range_inv_idx(update_range, linc+offsets[i], idx[i]);
    
          // Check if index is in the domain
          // Assumes update_range owns lower and upper edges of the domain
          for (int d=0; d<hdg->num_up_dirs; ++d) {
            int dir = hdg->update_dirs[d];
            if (idx[i][dir] < update_range->lower[dir] || idx[i][dir] > update_range->upper[dir]) {
              in_grid = 0;
            }
          }

          // If the index is in the domain, fetch the pointer
          // otherwise, point to the central cell
          if (in_grid) {
            fIn_d[i] = (const double*) gkyl_array_cfetch(fIn, linc + offsets[i]);
          }
          else {
            fIn_d[i] = (const double*) gkyl_array_cfetch(fIn, linc);
          }
          // reset in_grid for next neighbor value check
          in_grid = 1;
        }

        // Domain stencil location is handled by the kernel selectors
        // gen_surf_term contains both surf and boundary surf kernels
        hdg->equation->gen_surf_term(hdg->equation,
          dir1, dir2, xcc, hdg->grid.dx, idxc,
          keri, idx, fIn_d,
          (double*) gkyl_array_fetch(rhs, linc));
      }
    }
  }
}

// wrapper to call advance kernel on device
void
gkyl_hyper_dg_gen_stencil_advance_cu(gkyl_hyper_dg* hdg, const struct gkyl_range *update_range,
  const struct gkyl_array* GKYL_RESTRICT fIn, struct gkyl_array* GKYL_RESTRICT cflrate,
  struct gkyl_array* GKYL_RESTRICT rhs)
{
  int nblocks = update_range->nblocks;
  int nthreads = update_range->nthreads;

  gkyl_hyper_dg_gen_stencil_advance_cu_kernel<<<nblocks, nthreads>>>(hdg->on_dev, *update_range,
    fIn->on_dev, cflrate->on_dev, rhs->on_dev);
}

void
gkyl_hyper_dg_set_update_vol_cu(gkyl_hyper_dg *hdg, int update_vol_term)
{
  gkyl_hyper_dg_set_update_vol_cu_kernel<<<1,1>>>(hdg, update_vol_term);
}

gkyl_hyper_dg*
gkyl_hyper_dg_cu_dev_new(const struct gkyl_rect_grid *grid,
  const struct gkyl_basis *basis, const struct gkyl_dg_eqn *equation,
  int num_up_dirs, int update_dirs[GKYL_MAX_DIM], int zero_flux_flags[GKYL_MAX_DIM],
  int update_vol_term)
{
  gkyl_hyper_dg *up = (gkyl_hyper_dg*) gkyl_malloc(sizeof(gkyl_hyper_dg));

  up->ndim = basis->ndim;
  up->num_basis = basis->num_basis;
  up->num_up_dirs = num_up_dirs;
  up->grid = *grid;

  for (int i=0; i<num_up_dirs; ++i)
    up->update_dirs[i] = update_dirs[i];
  for (int i=0; i<GKYL_MAX_DIM; ++i)
    up->zero_flux_flags[i] = zero_flux_flags[i];
    
  up->update_vol_term = update_vol_term;

  // aquire pointer to equation object
  struct gkyl_dg_eqn *eqn = gkyl_dg_eqn_acquire(equation);
  up->equation = eqn->on_dev; // this is so the memcpy below has eqn on_dev

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);
  
  // copy host struct to device struct
  gkyl_hyper_dg *up_cu = (gkyl_hyper_dg*) gkyl_cu_malloc(sizeof(gkyl_hyper_dg));
  gkyl_cu_memcpy(up_cu, up, sizeof(struct gkyl_hyper_dg), GKYL_CU_MEMCPY_H2D);
  up->on_dev = up_cu; // set parent pointer

  up->equation = eqn; // updater should store host pointer

  return up;
}
