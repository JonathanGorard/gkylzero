#include "hip/hip_runtime.h"
/* -*- c++ -*- */

// CUB for reductions.
#include <hipcub/hipcub.hpp>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_util.h>
#include <gkyl_array_reduce.h>
}

__device__ static double
atomicMax_double(double* address, double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*) address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
      __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}

template <unsigned int BLOCKSIZE> __global__ void
arrayMax_blockRedAtomic_cub(const struct gkyl_array* inp, double* out)
{
  unsigned long linc = blockIdx.x*blockDim.x + threadIdx.x;

  // Specialize BlockReduce for type double.
  typedef hipcub::BlockReduce<double, BLOCKSIZE> BlockReduceT;

  // Allocate temporary storage in shared memory.
  __shared__ typename BlockReduceT::TempStorage temp;

  size_t nComp = inp->ncomp;
  size_t nCells = inp->size;

  const double *inp_d = (const double*) inp->data;

  for (size_t k = 0; k < nComp; ++k) {
    double f = -DBL_MAX;
    if (linc < nCells) f = inp_d[linc*nComp+k];
    double bResult = 0;
    bResult = BlockReduceT(temp).Reduce(f, hipcub::Max());
    if (threadIdx.x == 0)
      atomicMax_double(&out[k], bResult);
  }
}

template <unsigned int BLOCKSIZE>
__global__ void
arrayMax_range_blockRedAtomic_cub(const struct gkyl_array* inp, const struct gkyl_range range, double* out)
{
  unsigned long linc = blockIdx.x*blockDim.x + threadIdx.x;

  // Specialize BlockReduce for type double.
  typedef hipcub::BlockReduce<double, BLOCKSIZE> BlockReduceT;

  // Allocate temporary storage in shared memory.
  __shared__ typename BlockReduceT::TempStorage temp;

  long nCells = range.volume;
  size_t nComp = inp->ncomp;

  int idx[GKYL_MAX_DIM];

  for (size_t k = 0; k < nComp; ++k) {
    gkyl_sub_range_inv_idx(&range, linc, idx);
    long start = gkyl_range_idx(&range, idx);
    const double* fptr = (const double*) gkyl_array_cfetch(inp, start);
    double f = -DBL_MAX;
    if (linc < nCells) f = fptr[k];
    double bResult = 0;
    bResult = BlockReduceT(temp).Reduce(f, hipcub::Max());
    if (threadIdx.x == 0)
      atomicMax_double(&out[k], bResult);
  }
}

void
gkyl_array_reduce_max_cu(double *out_d, const struct gkyl_array* inp)
{
  const int nthreads = GKYL_DEFAULT_NUM_THREADS;  
  int nblocks = gkyl_int_div_up(inp->size, nthreads);
  arrayMax_blockRedAtomic_cub<nthreads><<<nblocks, nthreads>>>(inp->on_dev, out_d);
  // device synchronize required because out_d may be host pinned memory
  hipDeviceSynchronize();
}

void
gkyl_array_reduce_range_max_cu(double *out_d, const struct gkyl_array* inp, struct gkyl_range range)
{
  const int nthreads = GKYL_DEFAULT_NUM_THREADS;
  int nblocks = gkyl_int_div_up(range.volume, nthreads);
  arrayMax_range_blockRedAtomic_cub<nthreads><<<nblocks, nthreads>>>(inp->on_dev, range, out_d);
  // device synchronize required because out_d may be host pinned memory
  hipDeviceSynchronize();
}
