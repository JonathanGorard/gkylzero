#include "hip/hip_runtime.h"
/* -*- c++ -*- */

// CUB for reductions.
#include <hipcub/hipcub.hpp>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_util.h>
#include <gkyl_array_reduce.h>
}

__device__ static double
atomicMax_double(double* address, double val)
{
  unsigned long long int* address_as_ull = (unsigned long long int*) address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
      __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}

template <unsigned int BLOCKSIZE> __global__ void
arrayMax_blockRedAtomic_cub(const struct gkyl_array* inp, double* out)
{
  unsigned long linc = blockIdx.x*blockDim.x + threadIdx.x;

  // Specialize BlockReduce for type double.
  typedef hipcub::BlockReduce<double, BLOCKSIZE> BlockReduceT;

  // Allocate temporary storage in shared memory.
  __shared__ typename BlockReduceT::TempStorage temp;

  unsigned int nComp = inp->ncomp;
  unsigned long nCells = inp->size;

  const double *inp_d = (const double*) inp->data;

  for (unsigned int k = 0; k < nComp; ++k) {
    double f = inp_d[linc*nComp+k];
    double bResult;
    if (linc < nCells)
      bResult = BlockReduceT(temp).Reduce(f, hipcub::Max());
    if (threadIdx.x == 0)
      atomicMax_double(&out[k], bResult);
  };
}

template <unsigned int BLOCKSIZE>
__global__ void
arrayMax_range_blockRedAtomic_cub(const struct gkyl_array* inp, const struct gkyl_range range, double* out)
{
  unsigned long linc = blockIdx.x*blockDim.x + threadIdx.x;

  // Specialize BlockReduce for type double.
  typedef hipcub::BlockReduce<double, BLOCKSIZE> BlockReduceT;

  // Allocate temporary storage in shared memory.
  __shared__ typename BlockReduceT::TempStorage temp;

  long nCells = range.volume, nComp = inp->ncomp;

  int idx[GKYL_MAX_DIM];

  for (int k = 0; k < nComp; ++k) {
    gkyl_sub_range_inv_idx(&range, linc, idx);
    long start = gkyl_range_idx(&range, idx);
    double* fptr = (double*) gkyl_array_cfetch(inp, start);
    double f = fptr[k];
    double bResult;
    if (linc < nCells)
      bResult = BlockReduceT(temp).Reduce(f, hipcub::Max());
    if (threadIdx.x == 0)
      atomicMax_double(&out[k], bResult);
  };
}

void
gkyl_array_reduce_max_cu(double *out_d, const struct gkyl_array* inp)
{
  int numCells = inp->size;
  const int blockSize = GKYL_DEFAULT_NUM_THREADS;
  arrayMax_blockRedAtomic_cub<blockSize><<<gkyl_int_div_up(numCells, blockSize), blockSize>>>(inp->on_device, out_d);
  hipDeviceSynchronize();
}

void
gkyl_array_reduce_range_max_cu(double *out_d, const struct gkyl_array* inp, struct gkyl_range range)
{
  int numCells = range.volume;
  const int blockSize = GKYL_DEFAULT_NUM_THREADS;
  arrayMax_range_blockRedAtomic_cub<blockSize><<<gkyl_int_div_up(numCells, blockSize), blockSize>>>(
    inp->on_device, range, out_d);
}
