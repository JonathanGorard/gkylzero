#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_rad_gyrokinetic_drag.h>    
#include <gkyl_dg_rad_gyrokinetic_drag_priv.h>
}

#include <cassert>

// CUDA kernel to set pointer to auxiliary fields.
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_rad_gyrokinetic_drag_set_auxfields_cu_kernel(const struct gkyl_dg_eqn *eqn, 
  const struct gkyl_array *nvnu_sum, const struct gkyl_array *nvsqnu_sum)
{
  struct dg_rad_gyrokinetic_drag *rad_gyrokinetic_drag = container_of(eqn, struct dg_rad_gyrokinetic_drag, eqn);
  rad_gyrokinetic_drag->auxfields.nvnu_sum = nvnu_sum;
  rad_gyrokinetic_drag->auxfields.nvsqnu_sum = nvsqnu_sum;
}

// Host-side wrapper for set_auxfields_cu_kernel
void
gkyl_rad_gyrokinetic_drag_set_auxfields_cu(const struct gkyl_dg_eqn *eqn, struct gkyl_dg_rad_gyrokinetic_drag_auxfields auxin)
{
  gkyl_rad_gyrokinetic_drag_set_auxfields_cu_kernel<<<1,1>>>(eqn, 
  auxin.nvnu_sum->on_dev, auxin.nvsqnu_sum->on_dev);
}

// CUDA kernel to set device pointers to range object and rad_gyrokinetic_drag kernel function
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_rad_gyrokinetic_drag_set_cu_dev_ptrs(struct dg_rad_gyrokinetic_drag *rad_gyrokinetic_drag, enum gkyl_basis_type b_type,
  int cv_index, int cdim, int vdim, int poly_order)
{
  rad_gyrokinetic_drag->auxfields.nvnu_sum = 0; 
  rad_gyrokinetic_drag->auxfields.nvsqnu_sum = 0; 

  rad_gyrokinetic_drag->eqn.surf_term = surf;
  rad_gyrokinetic_drag->eqn.boundary_surf_term = boundary_surf;

  const gkyl_dg_rad_gyrokinetic_drag_vol_kern_list *vol_kernels;
  const gkyl_dg_rad_gyrokinetic_drag_surf_kern_list *surf_vpar_kernels, *surf_mu_kernels;
  const gkyl_dg_rad_gyrokinetic_drag_boundary_surf_kern_list *boundary_surf_vpar_kernels, *boundary_surf_mu_kernels;
  
  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      vol_kernels = ser_vol_kernels;
      surf_vpar_kernels = ser_surf_vpar_kernels;
      surf_mu_kernels = ser_surf_mu_kernels;
      boundary_surf_vpar_kernels = ser_boundary_surf_vpar_kernels;
      boundary_surf_mu_kernels = ser_boundary_surf_mu_kernels;
      break;

    default:
      assert(false);
      break;    
  }  
  rad_gyrokinetic_drag->eqn.vol_term = vol_kernels[cv_index].kernels[poly_order];

  rad_gyrokinetic_drag->surf[0] = surf_vpar_kernels[cv_index].kernels[poly_order];
  if (vdim>1)
    rad_gyrokinetic_drag->surf[1] = surf_mu_kernels[cv_index].kernels[poly_order];

  rad_gyrokinetic_drag->boundary_surf[0] = boundary_surf_vpar_kernels[cv_index].kernels[poly_order];
  if (vdim>1)
    rad_gyrokinetic_drag->boundary_surf[1] = boundary_surf_mu_kernels[cv_index].kernels[poly_order];
}

struct gkyl_dg_eqn*
gkyl_dg_rad_gyrokinetic_drag_cu_dev_new(const struct gkyl_basis* conf_basis, 
  const struct gkyl_basis* phase_basis, const struct gkyl_range *phase_range){
  struct dg_rad_gyrokinetic_drag *rad_gyrokinetic_drag = (struct dg_rad_gyrokinetic_drag*) gkyl_malloc(sizeof(struct dg_rad_gyrokinetic_drag));

  int cdim = conf_basis->ndim, pdim = phase_basis->ndim, vdim = pdim-cdim;
  int poly_order = conf_basis->poly_order;
  
  rad_gyrokinetic_drag->cdim = cdim;
  rad_gyrokinetic_drag->pdim = pdim;
  rad_gyrokinetic_drag->phase_range = *phase_range;

  rad_gyrokinetic_drag->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(rad_gyrokinetic_drag->eqn.flags);
  rad_gyrokinetic_drag->eqn.ref_count = gkyl_ref_count_init(gkyl_rad_gyrokinetic_drag_free);

  // copy the host struct to device struct
  struct dg_rad_gyrokinetic_drag *rad_gyrokinetic_drag_cu = (struct dg_rad_gyrokinetic_drag*) gkyl_cu_malloc(sizeof(struct dg_rad_gyrokinetic_drag));
  gkyl_cu_memcpy(rad_gyrokinetic_drag_cu, rad_gyrokinetic_drag, sizeof(struct dg_rad_gyrokinetic_drag), GKYL_CU_MEMCPY_H2D);

  dg_rad_gyrokinetic_drag_set_cu_dev_ptrs<<<1,1>>>(rad_gyrokinetic_drag_cu, conf_basis->b_type, cv_index[cdim].vdim[vdim],
    cdim, vdim, poly_order);

  // set parent on_dev pointer
  rad_gyrokinetic_drag->eqn.on_dev = &rad_gyrokinetic_drag_cu->eqn;

  return &rad_gyrokinetic_drag->eqn;
}
