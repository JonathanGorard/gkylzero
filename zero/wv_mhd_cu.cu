#include "hip/hip_runtime.h"
extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_wv_mhd.h>
#include <gkyl_wv_mhd_priv.h>
}

// CUDA kernel to set device pointers to mhd kernel functions
__global__ static void
wv_mhd_set_cu_dev_ptrs(struct wv_mhd *mhd)
{
  mhd->eqn.waves_func = wave_roe;
  mhd->eqn.qfluct_func = qfluct_roe;
  mhd->eqn.max_speed_func = max_speed;
  mhd->eqn.rotate_to_local_func = rot_to_local_rect;
  mhd->eqn.rotate_to_global_func = rot_to_global_rect;
}

// CPU interface to create and track a GPU object
struct gkyl_wv_eqn*
gkyl_wv_mhd_cu_dev_new(double gas_gamma, const char *divergence_constraint)
{
  // STEP 0. CREATE OBJECT ON CPU
  struct wv_mhd *mhd = (struct wv_mhd*) gkyl_malloc(sizeof(struct wv_mhd));

  // STEP 1. SET PRIMITIVE DATA IN THE CPU OBJECT
  // These data are either primitive data (numbers and flags) that can be
  // used by GPU directly, or functions not used by GPU
  mhd->eqn.type = GKYL_EQN_MHD;
  mhd->gas_gamma = gas_gamma;

  if (strcmp(divergence_constraint, "none") == 0) {
    mhd->divergence_constraint = DIVB_NONE;
    mhd->eqn.num_equations = 8;
    mhd->eqn.num_waves = 7;
  } else if (strcmp(divergence_constraint, "eight_waves") == 0) {
    mhd->divergence_constraint = DIVB_EIGHT_WAVES;
    mhd->eqn.num_equations = 8;
    mhd->eqn.num_waves = 7;  // entropy and divB waves are merged into one wave
  } else if (strcmp(divergence_constraint, "glm") == 0) {
    mhd->divergence_constraint = DIVB_GLM;
    mhd->eqn.num_equations = 9;
    mhd->eqn.num_waves = 9;
    mhd->eqn.rotate_to_local_func = rot_to_local_rect_glm;
    mhd->eqn.rotate_to_global_func = rot_to_global_rect_glm;
  } else { // TODO: Warn or throw an error
    mhd->divergence_constraint = DIVB_NONE;
    mhd->eqn.num_equations = 8;
    mhd->eqn.num_waves = 7;
  }

  mhd->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(mhd->eqn.flags);

  mhd->eqn.ref_count = gkyl_ref_count_init(gkyl_wv_mhd_free);

  // STEP 2. COPY HOST OBJECT TO DEVICE OBJECT
  struct wv_mhd *mhd_cu = (struct wv_mhd*) gkyl_cu_malloc(sizeof(struct wv_mhd));
  gkyl_cu_memcpy(mhd_cu, mhd, sizeof(struct wv_mhd), GKYL_CU_MEMCPY_H2D);

  // STEP 3. SET DEVICE MEMBER FUNCTION POINTERS IN DEVICE OBJECT
  wv_mhd_set_cu_dev_ptrs<<<1,1>>>(mhd_cu);

  /* STEP 4. KEEP POINTER TO THE DEVICE OBJECT */
  mhd->eqn.on_dev = &mhd_cu->eqn;

  return &mhd->eqn;
}
