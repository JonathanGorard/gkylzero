#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_vlasov_lte_correct.h>
#include <gkyl_vlasov_lte_correct_priv.h>
#include <gkyl_range.h>
}

__global__ static void
gkyl_vlasov_lte_correct_all_moments_abs_diff_cu_ker(struct gkyl_range conf_range, 
  int vdim, int nc, 
  const struct gkyl_array *moms_target, const struct gkyl_array *moms_iter, 
  struct gkyl_array *abs_diff_moms)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *moms_local = (const double*) gkyl_array_cfetch(moms_iter, loc);
    const double *moms_target_local = (const double*) gkyl_array_cfetch(moms_target, loc);
    double *abs_diff_moms_local = (double*) gkyl_array_fetch(abs_diff_moms, loc);
    // Compute the absolute value of the difference of cell averages 
    // Note: max error found by follow-up thread-safe reduction operation
    for (int d=0; d<vdim+2; ++d) {
      abs_diff_moms_local[d] = fabs(moms_local[d*nc] - moms_target_local[d*nc]);
    }
  }
}

void
gkyl_vlasov_lte_correct_all_moments_abs_diff_cu(const struct gkyl_range *conf_range, 
  int vdim, int nc, 
  const struct gkyl_array *moms_target, const struct gkyl_array *moms_iter, 
  struct gkyl_array *moms_abs_diff)
{
  int nblocks = conf_range->nblocks, nthreads = conf_range->nthreads;
  gkyl_vlasov_lte_correct_all_moments_abs_diff_cu_ker<<<nblocks, nthreads>>>(*conf_range, 
    vdim, nc, moms_target->on_dev, moms_iter->on_dev, moms_abs_diff->on_dev);
}