#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_vlasov.h>    
#include <gkyl_dg_vlasov_priv.h>
}

#include <cassert>

// CUDA kernel to set pointer to qmem = q/m*EM
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_vlasov_set_qmem_cu_kernel(const struct gkyl_dg_eqn *eqn, const struct gkyl_array *qmem)
{
  struct dg_vlasov *vlasov = container_of(eqn, struct dg_vlasov, eqn);
  vlasov->qmem = qmem;
}

// Host-side wrapper for set_qmem_cu_kernel
void
gkyl_vlasov_set_qmem_cu(const struct gkyl_dg_eqn *eqn, const struct gkyl_array *qmem)
{
  gkyl_vlasov_set_qmem_cu_kernel<<<1,1>>>(eqn, qmem->on_dev);
}

// CUDA kernel to set device pointers to range object and vlasov kernel function
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_vlasov_set_cu_dev_ptrs(struct dg_vlasov *vlasov, enum gkyl_basis_type b_type,
  int cv_index, int cdim, int vdim, int poly_order, enum gkyl_field_id field_id)
{
  vlasov->qmem = 0; 

  vlasov->eqn.vol_term = vol;
  vlasov->eqn.surf_term = surf;
  vlasov->eqn.boundary_surf_term = boundary_surf;

  const gkyl_dg_vlasov_stream_vol_kern_list *stream_vol_kernels;
  const gkyl_dg_vlasov_vol_kern_list *vol_kernels;
  const gkyl_dg_vlasov_stream_surf_kern_list *stream_surf_x_kernels, *stream_surf_y_kernels, *stream_surf_z_kernels;
  const gkyl_dg_vlasov_accel_surf_kern_list *accel_surf_vx_kernels, *accel_surf_vy_kernels, *accel_surf_vz_kernels;
  const gkyl_dg_vlasov_accel_boundary_surf_kern_list *accel_boundary_surf_vx_kernels, *accel_boundary_surf_vy_kernels,
    *accel_boundary_surf_vz_kernels;
  
  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      stream_vol_kernels = ser_stream_vol_kernels;
      vol_kernels = ser_vol_kernels;
      stream_surf_x_kernels = ser_stream_surf_x_kernels;
      stream_surf_y_kernels = ser_stream_surf_y_kernels;
      stream_surf_z_kernels = ser_stream_surf_z_kernels;
      accel_surf_vx_kernels = ser_accel_surf_vx_kernels;
      accel_surf_vy_kernels = ser_accel_surf_vy_kernels;
      accel_surf_vz_kernels = ser_accel_surf_vz_kernels;
      accel_boundary_surf_vx_kernels = ser_accel_boundary_surf_vx_kernels;
      accel_boundary_surf_vy_kernels = ser_accel_boundary_surf_vy_kernels;
      accel_boundary_surf_vz_kernels = ser_accel_boundary_surf_vz_kernels;
      
      break;

    case GKYL_BASIS_MODAL_TENSOR:
      stream_vol_kernels = ten_stream_vol_kernels;
      vol_kernels = ten_vol_kernels;
      stream_surf_x_kernels = ten_stream_surf_x_kernels;
      stream_surf_y_kernels = ten_stream_surf_y_kernels;
      stream_surf_z_kernels = ten_stream_surf_z_kernels;
      accel_surf_vx_kernels = ten_accel_surf_vx_kernels;
      accel_surf_vy_kernels = ten_accel_surf_vy_kernels;
      accel_surf_vz_kernels = ten_accel_surf_vz_kernels;
      accel_boundary_surf_vx_kernels = ten_accel_boundary_surf_vx_kernels;
      accel_boundary_surf_vy_kernels = ten_accel_boundary_surf_vy_kernels;
      accel_boundary_surf_vz_kernels = ten_accel_boundary_surf_vz_kernels;
      break;

    default:
      assert(false);
      break;    
  }  
  if (field_id == GKYL_FIELD_NULL)
    vlasov->vol = stream_vol_kernels[cv_index].kernels[poly_order];
  else
    vlasov->vol = vol_kernels[cv_index].kernels[poly_order];

  vlasov->stream_surf[0] = stream_surf_x_kernels[cv_index].kernels[poly_order];
  if (cdim>1)
    vlasov->stream_surf[1] = stream_surf_y_kernels[cv_index].kernels[poly_order];
  if (cdim>2)
    vlasov->stream_surf[2] = stream_surf_z_kernels[cv_index].kernels[poly_order];

  vlasov->accel_surf[0] = accel_surf_vx_kernels[cv_index].kernels[poly_order];
  if (vdim>1)
    vlasov->accel_surf[1] = accel_surf_vy_kernels[cv_index].kernels[poly_order];
  if (vdim>2)
    vlasov->accel_surf[2] = accel_surf_vz_kernels[cv_index].kernels[poly_order];

  vlasov->accel_boundary_surf[0] = accel_boundary_surf_vx_kernels[cv_index].kernels[poly_order];
  if (vdim>1)
    vlasov->accel_boundary_surf[1] = accel_boundary_surf_vy_kernels[cv_index].kernels[poly_order];
  if (vdim>2)
    vlasov->accel_boundary_surf[2] = accel_boundary_surf_vz_kernels[cv_index].kernels[poly_order];
}

struct gkyl_dg_eqn*
gkyl_dg_vlasov_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_basis* pbasis,
  const struct gkyl_range* conf_range, enum gkyl_field_id field_id)
{
  struct dg_vlasov *vlasov = (struct dg_vlasov*) gkyl_malloc(sizeof(struct dg_vlasov));

  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  vlasov->cdim = cdim;
  vlasov->pdim = pdim;

  vlasov->eqn.num_equations = 1;
  vlasov->conf_range = *conf_range;

  vlasov->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(vlasov->eqn.flags);
  vlasov->eqn.ref_count = gkyl_ref_count_init(gkyl_vlasov_free);

  // copy the host struct to device struct
  struct dg_vlasov *vlasov_cu = (struct dg_vlasov*) gkyl_cu_malloc(sizeof(struct dg_vlasov));
  gkyl_cu_memcpy(vlasov_cu, vlasov, sizeof(struct dg_vlasov), GKYL_CU_MEMCPY_H2D);

  dg_vlasov_set_cu_dev_ptrs<<<1,1>>>(vlasov_cu, cbasis->b_type, cv_index[cdim].vdim[vdim],
    cdim, vdim, poly_order, field_id);

  // set parent on_dev pointer
  vlasov->eqn.on_dev = &vlasov_cu->eqn;
  
  return &vlasov->eqn;
}
