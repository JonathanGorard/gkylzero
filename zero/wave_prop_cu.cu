#include "hip/hip_runtime.h"
extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_rect_decomp.h>
#include <gkyl_wave_geom.h>
#include <gkyl_wave_geom_priv.h>
#include <gkyl_wave_prop.h>
#include <gkyl_wave_prop_priv.h>
}

GKYL_CU_D static void limit_waves_cu(const gkyl_wave_prop *wv,
                                     const struct gkyl_range *slice_range,
                                     int lower, int upper,
                                     struct gkyl_array *waves,
                                     const struct gkyl_array *speed) {
  int meqn = wv->equation->num_equations, mwave = wv->equation->num_waves;

  for (int mw = 0; mw < mwave; ++mw) {
    const double *wl = (const double *)gkyl_array_cfetch(
        waves, gkyl_ridx(*slice_range, lower - 1));
    const double *wr = (const double *)gkyl_array_cfetch(
        waves, gkyl_ridx(*slice_range, lower));

    double dotr = wave_dot_prod(meqn, &wl[mw * meqn], &wr[mw * meqn]);

    for (int i = lower; i <= upper; ++i) {
      double dotl = dotr;

      double *GKYL_RESTRICT wi =
          (double *)gkyl_array_fetch(waves, gkyl_ridx(*slice_range, i));
      const double *GKYL_RESTRICT wi1 =
          (double *)gkyl_array_cfetch(waves, gkyl_ridx(*slice_range, i + 1));

      double wnorm2 = wave_dot_prod(meqn, &wi[mw * meqn], &wi[mw * meqn]);
      dotr = wave_dot_prod(meqn, &wi[mw * meqn], &wi1[mw * meqn]);

      if (wnorm2 > 0) {
        const double *s = (const double *)gkyl_array_cfetch(
            speed, gkyl_ridx(*slice_range, i));
        double r = s[mw] > 0 ? dotl / wnorm2 : dotr / wnorm2;
        double theta = limiter_function(r, wv->limiter);
        wave_rescale(meqn, theta, &wi[mw * meqn]);
      }
    }
  }
}

// CPU interface to create and track a GPU object
gkyl_wave_prop *gkyl_wave_prop_cu_dev_new(gkyl_wave_prop_inp winp) {
  // STEP: CREATE HOST OBJECT
  gkyl_wave_prop *up = (gkyl_wave_prop *)gkyl_malloc(sizeof(gkyl_wave_prop));

  // STEP: SET HOST OR COMMON HOST/DEVICE DATA IN HOST OBJECT
  up->grid = *(winp.grid);
  up->ndim = up->grid.ndim;
  up->num_up_dirs = winp.num_up_dirs;
  for (int i = 0; i < winp.num_up_dirs; ++i)
    up->update_dirs[i] = winp.update_dirs[i];
  up->limiter = winp.limiter == 0 ? GKYL_MONOTONIZED_CENTERED : winp.limiter;
  up->cfl = winp.cfl;
  int nghost[3] = {2, 2, 2};
  struct gkyl_range range, ext_range;
  gkyl_create_grid_ranges(&up->grid, nghost, &ext_range, &range);

  up->equation = winp.equation->on_dev;
  up->geom = winp.geom->on_dev;

  // force null pointers that can be handled by gkyl_array_release
  up->waves = NULL;
  up->speeds = NULL;
  up->flux2 = NULL;

  // STEP: COPY HOST OBJECT TO DEVICE OBJECT
  gkyl_wave_prop *up_dev =
      (gkyl_wave_prop *)gkyl_cu_malloc(sizeof(gkyl_wave_prop));
  gkyl_cu_memcpy(up_dev, up, sizeof(gkyl_wave_prop), GKYL_CU_MEMCPY_H2D);

  up->equation = gkyl_wv_eqn_acquire(winp.equation);
  up->geom = gkyl_wave_geom_acquire(winp.geom);

  // STEP: SET DEVICE DATA

  // STEP: KEEP POINTER TO THE DEVICE OBJECT
  up->on_dev = up_dev;

  return up;
}

__global__ void do_gkyl_wave_prop_cu_dev_advance(
    const gkyl_wave_prop *wv, double tm, double dt,
    const struct gkyl_range update_range, const struct gkyl_array *qin,
    struct gkyl_array *qout, struct gkyl_wave_prop_status *status) {
  int ndim = update_range.ndim;
  int idxl[3], idxc[3], idxr[3];

  // int meqn = wv->equation->num_equations, mwave = wv->equation->num_waves;
  // FIXME
  const int meqn = 8;
  const int mwave = 6;

  double cfla = 0.0, cfl = wv->cfl, cflm = 1.1 * cfl;

  double ql_local[meqn], qr_local[meqn];
  double waves_local[meqn * mwave];
  double delta[meqn], amdq[meqn], apdq[meqn];

  // assign buffers for each thread to solve RP on four edges, and computing
  // fluxes on two edges for updating one cell
  extern __shared__ double dummy[];
  int base = 0;

  double *waves = dummy + base + (meqn * mwave * 4) * (threadIdx.x);
  base += (meqn * mwave * 4) * (blockDim.x);

  double *speeds = dummy + base + (mwave * 4) * (threadIdx.x);
  base += (mwave * 4) * (blockDim.x);

  double *flux2 = dummy + base + (meqn * 2) * (threadIdx.x);
  base += (meqn * 2) * (blockDim.x);

  // iterate over all cells, one thread updates one cell a time */
  for (unsigned long linc1 = threadIdx.x + blockIdx.x * blockDim.x;
       linc1 < update_range.volume; linc1 += blockDim.x * gridDim.x) {
    // inverse index from linc1 to idxc
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idxc={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&update_range, linc1, idxc);

    gkyl_copy_int_arr(ndim, idxc, idxl);
    gkyl_copy_int_arr(ndim, idxc, idxr);

    for (int d = 0; d < wv->num_up_dirs; ++d) {
      int dir = wv->update_dirs[d];

      double dtdx = dt / wv->grid.dx[dir];

      /****************************************/
      /* SOLVE RIEMANN PROBLEMS ON FOUR EDGES */
      /****************************************/
      for (int i = 0; i <= 3; ++i) {
        idxl[dir] += i - 2;
        idxr[dir] += i - 1; // left and right cells of the edge

        const struct gkyl_wave_cell_geom *cg =
            gkyl_wave_geom_get(wv->geom, idxr);

        long lidx = gkyl_range_idx(&update_range, idxl);
        long ridx = gkyl_range_idx(&update_range, idxr);

        const double *qinl = (const double *)gkyl_array_cfetch(qin, lidx);
        const double *qinr = (const double *)gkyl_array_cfetch(qin, ridx);

        wv->equation->rotate_to_local_func(cg->tau1[dir], cg->tau2[dir],
                                           cg->norm[dir], qinl, ql_local);
        wv->equation->rotate_to_local_func(cg->tau1[dir], cg->tau2[dir],
                                           cg->norm[dir], qinr, qr_local);

        calc_jump(meqn, ql_local, qr_local, delta);
        double *s = speeds + mwave * i;
        wv->equation->waves_func(wv->equation, delta, ql_local, qr_local,
                                 waves_local, s);

        double lenr = cg->lenr[dir];
        double *my_waves = waves + mwave * meqn * i;
        for (int mw = 0; mw < mwave; ++mw) {
          wv->equation->rotate_to_global_func(
              cg->tau1[dir], cg->tau2[dir], cg->norm[dir],
              &waves_local[mw * meqn], &my_waves[mw * meqn]);
          s[mw] *= lenr;
        }

        wv->equation->qfluct_func(wv->equation, qinl, qinr, my_waves, s, amdq,
                                  apdq);

        double *qoutl = (double *)gkyl_array_fetch(qout, lidx);
        double *qoutr = (double *)gkyl_array_fetch(qout, ridx);

        calc_first_order_update(meqn, dtdx / cg->kappa, qoutl, qoutr, amdq,
                                apdq);
        cfla = calc_cfla(mwave, cfla, dtdx / cg->kappa, s);
      }

      if (cfla > cflm) {
        status->success = 0;
        status->dt_suggested = dt * cfl / cfla;
        return;
      }

      /****************************/
      /* LIMIT WAVES ON TWO EDGES */
      /****************************/

      //   limit_waves_cu(wv, &slice_range, update_range.lower[dir],
      //      update_range.upper[dir]+1, waves, wv->speeds);

      /*******************************************************************/
      /* COMPUTE 2ND-ORDER FLUXES ON LEFT AND RIGHT EDGES OF TARGET CELL */
      /*******************************************************************/

      for (int i = 0; i < meqn * 2; ++i)
        flux2[i] = 0.;

      idxl[dir] = idxc[dir] - 1;
      const struct gkyl_wave_cell_geom *cg = gkyl_wave_geom_get(wv->geom, idxl);
      double kappal = cg->kappa;

      for (int i = 1; i <= 2; ++i) {
        const double *my_waves = waves + mwave * meqn * i;
        const double *s = speeds + mwave * i;

        // we stored flux2 on two edges only, thus the shift is i-1 not i
        double *my_flux2 = flux2 + meqn * (i - 1);

        idxl[dir] = idxc[dir] + i - 1;

        const struct gkyl_wave_cell_geom *cg =
            gkyl_wave_geom_get(wv->geom, idxl);
        double kappar = cg->kappa;

        for (int mw = 0; mw < mwave; ++mw)
          calc_second_order_flux(meqn, dtdx / (0.5 * (kappal + kappar)), s[mw],
                                 &my_waves[mw * meqn], my_flux2);

        kappal = kappar;
      }

      /*********************************************/
      /* ADD 2ND-ORDER CORRECTION ONTO TARGET CELL */
      /*********************************************/
      long linc = gkyl_range_idx(&update_range, idxc);
      double *qc = (double *)gkyl_array_fetch(qout, linc);
      cg = gkyl_wave_geom_get(wv->geom, idxc);
      calc_second_order_update(meqn, dtdx / cg->kappa, qc, flux2, flux2 + meqn);
    }
  }

  double dt_suggested = dt * cfl / fmax(cfla, DBL_MIN);
  status->dt_suggested = dt_suggested > dt ? dt_suggested : dt;
  status->success = 1;
}

struct gkyl_wave_prop_status
gkyl_wave_prop_cu_dev_advance(const gkyl_wave_prop *wv, double tm, double dt,
                              const struct gkyl_range *update_range,
                              const struct gkyl_array *qin,
                              struct gkyl_array *qout) {
  int nthreads = update_range->nthreads;
  int nblocks = update_range->nblocks;

  int meqn = wv->equation->num_equations, mwave = wv->equation->num_waves;
  int shared_mem_size = 0;
  shared_mem_size += (meqn * mwave * 4) * (nthreads);
  shared_mem_size += (mwave * 4) * (nthreads);
  shared_mem_size += (meqn * 2) * (nthreads);
  shared_mem_size *= sizeof(double);

  struct gkyl_wave_prop_status *status_dev =
      (struct gkyl_wave_prop_status *)gkyl_cu_malloc(
          sizeof(struct gkyl_wave_prop_status));

  gkyl_array_copy(qout, qin);
  do_gkyl_wave_prop_cu_dev_advance<<<nthreads, nblocks, shared_mem_size>>>(
      wv->on_dev, tm, dt, *update_range, qin->on_dev, qout->on_dev, status_dev);
  checkCuda(hipGetLastError());

  struct gkyl_wave_prop_status status;
  gkyl_cu_memcpy(&status, status_dev, sizeof(struct gkyl_wave_prop_status),
                 GKYL_CU_MEMCPY_D2H);
  gkyl_cu_free(status_dev);

  return status;
}
