#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_euler_iso.h>    
#include <gkyl_dg_euler_iso_priv.h>
}

#include <cassert>

#define CK(lst,cdim,poly_order) lst[cdim-1].kernels[poly_order]

// CUDA kernel to set pointer to auxiliary fields.
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_euler_iso_set_auxfields_cu_kernel(const struct gkyl_dg_eqn *eqn, const struct gkyl_array *u_i)
{
  struct dg_euler_iso *euler_iso = container_of(eqn, struct dg_euler_iso, eqn);
  euler_iso->auxfields.u_i = u_i;
}

// Host-side wrapper for set_auxfields_cu_kernel
void
gkyl_euler_iso_set_auxfields_cu(const struct gkyl_dg_eqn *eqn, struct gkyl_dg_euler_iso_auxfields auxin)
{
  gkyl_euler_iso_set_auxfields_cu_kernel<<<1,1>>>(eqn, auxin.u_i->on_dev);
}

__global__ void static
dg_euler_iso_set_cu_dev_ptrs(struct dg_euler_iso* euler_iso, enum gkyl_basis_type b_type, int cdim, int poly_order)
{
  euler_iso->auxfields.u_i = 0; 

  const gkyl_dg_euler_iso_vol_kern_list *vol_kernels;
  const gkyl_dg_euler_iso_surf_kern_list *surf_x_kernels, *surf_y_kernels, *surf_z_kernels;  
  
  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      vol_kernels = ser_vol_kernels;
      surf_x_kernels = ser_surf_x_kernels;
      surf_y_kernels = ser_surf_y_kernels;
      surf_z_kernels = ser_surf_z_kernels;
      break;

    default:
      assert(false);
      break;    
  }  
  
  euler_iso->eqn.vol_term = vol;
  euler_iso->eqn.surf_term = surf;
  euler_iso->eqn.boundary_surf_term = boundary_surf;

  euler_iso->vol =  CK(vol_kernels, cdim, poly_order);

  euler_iso->surf[0] = CK(surf_x_kernels, cdim, poly_order);
  if (cdim>1)
    euler_iso->surf[1] = CK(surf_y_kernels, cdim, poly_order);
  if (cdim>2)
    euler_iso->surf[2] = CK(surf_z_kernels, cdim, poly_order);
}

struct gkyl_dg_eqn*
gkyl_dg_euler_iso_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_range* conf_range, 
  double vth)
{
  struct dg_euler_iso *euler_iso = (struct dg_euler_iso*) gkyl_malloc(sizeof(struct dg_euler_iso));

  // set basic parameters
  euler_iso->eqn.num_equations = 4;
  euler_iso->vth = vth;

  euler_iso->conf_range = *conf_range;

  euler_iso->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(euler_iso->eqn.flags);
  euler_iso->eqn.ref_count = gkyl_ref_count_init(gkyl_euler_iso_free);

  // copy the host struct to device struct
  struct dg_euler_iso *euler_iso_cu = (struct dg_euler_iso*) gkyl_cu_malloc(sizeof(struct dg_euler_iso));
  gkyl_cu_memcpy(euler_iso_cu, euler_iso, sizeof(struct dg_euler_iso), GKYL_CU_MEMCPY_H2D);
  dg_euler_iso_set_cu_dev_ptrs<<<1,1>>>(euler_iso_cu, cbasis->b_type, cbasis->ndim, cbasis->poly_order);

  // set parent on_dev pointer
  euler_iso->eqn.on_dev = &euler_iso_cu->eqn;

  return &euler_iso->eqn;
}
