#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_diffusion.h>    
#include <gkyl_dg_diffusion_priv.h>
}

#include <cassert>

#define CK(lst,cdim,poly_order) lst[cdim-1].kernels[poly_order]

// CUDA kernel to set pointer to auxiliary fields.
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_diffusion_set_auxfields_cu_kernel(const struct gkyl_dg_eqn* eqn, const struct gkyl_array* D)
{
  struct dg_diffusion* diffusion = container_of(eqn, struct dg_diffusion, eqn);
  diffusion->auxfields.D = D;
}

// Host-side wrapper for set_auxfields_cu_kernel
void
gkyl_diffusion_set_auxfields_cu(const struct gkyl_dg_eqn* eqn, struct gkyl_dg_diffusion_auxfields auxin)
{
  gkyl_diffusion_set_auxfields_cu_kernel<<<1,1>>>(eqn, auxin.D->on_dev);
}

__global__ void static
dg_diffusion_set_cu_dev_ptrs(struct dg_diffusion* diffusion, enum gkyl_basis_type b_type, int cdim, int poly_order)
{
  diffusion->auxfields.D = 0; 

  const gkyl_dg_diffusion_vol_kern_list* vol_kernels;
  const gkyl_dg_diffusion_surf_kern_list* surf_x_kernels;
  const gkyl_dg_diffusion_surf_kern_list* surf_y_kernels;
  const gkyl_dg_diffusion_surf_kern_list* surf_z_kernels; 

  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      vol_kernels = ser_vol_kernels;
      surf_x_kernels = ser_surf_x_kernels;
      surf_y_kernels = ser_surf_y_kernels;
      surf_z_kernels = ser_surf_z_kernels;
      break;

    default:
      assert(false);
      break;    
  } 
  
  diffusion->eqn.surf_term = surf;
  //advection->eqn.boundary_surf_term = boundary_surf;

  diffusion->eqn.vol_term = CK(vol_kernels, cdim, poly_order);

  diffusion->surf[0] = CK(surf_x_kernels, cdim, poly_order);
  if (cdim>1)
    diffusion->surf[1] = CK(surf_y_kernels, cdim, poly_order);
  if (cdim>2)
    diffusion->surf[2] = CK(surf_z_kernels, cdim, poly_order);
}

struct gkyl_dg_eqn*
gkyl_dg_diffusion_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_range* conf_range)
{
  struct dg_diffusion* diffusion = (struct dg_diffusion*) gkyl_malloc(sizeof(struct dg_diffusion));

  // set basic parameters
  diffusion->eqn.num_equations = 1;
  diffusion->conf_range = *conf_range;

  diffusion->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(diffusion->eqn.flags);
  diffusion->eqn.ref_count = gkyl_ref_count_init(gkyl_diffusion_free);

  // copy the host struct to device struct
  struct dg_diffusion* diffusion_cu = (struct dg_diffusion*) gkyl_cu_malloc(sizeof(struct dg_diffusion));
  gkyl_cu_memcpy(diffusion_cu, diffusion, sizeof(struct dg_diffusion), GKYL_CU_MEMCPY_H2D);
  dg_diffusion_set_cu_dev_ptrs<<<1,1>>>(diffusion_cu, cbasis->b_type, cbasis->ndim, cbasis->poly_order);

  // set parent on_dev pointer
  diffusion->eqn.on_dev = &diffusion_cu->eqn;

  return &diffusion->eqn;
}
