#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_gyrokinetic_maxwellian_correct.h>
#include <gkyl_gyrokinetic_maxwellian_correct_priv.h>
#include <gkyl_range.h>
}

__global__ static void
gkyl_gyrokinetic_maxwellian_correct_all_moments_abs_diff_cu_ker(struct gkyl_range conf_range, 
  int nc, const struct gkyl_array *moms_target, const struct gkyl_array *moms_iter, 
  struct gkyl_array *abs_diff_moms)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *moms_local = (const double*) gkyl_array_cfetch(moms_iter, loc);
    const double *moms_target_local = (const double*) gkyl_array_cfetch(moms_target, loc);
    double *abs_diff_moms_local = (double*) gkyl_array_fetch(abs_diff_moms, loc);
    // Compute the absolute value of the difference of cell averages 
    // Note: max error found by follow-up thread-safe reduction operation
    // Also: for density and temperature, this error is a relative error compared to the target moment value
    // so that we can converge to the correct target moments in SI units and minimize finite precision issues.
    abs_diff_moms_local[0] = fabs(moms_local[0*nc] - moms_target_local[0*nc])/moms_target_local[0*nc];
    abs_diff_moms_local[2] = fabs(moms_local[2*nc] - moms_target_local[2*nc])/moms_target_local[2*nc];
    // However, u_par may be ~ 0 and if it is, we need to use absolute error. We can converge safely using
    // absolute error if u_par ~ O(1). Otherwise, we use relative error for u_par. 
    if (fabs(moms_target_local[1*nc]) < 1.0) {
      abs_diff_moms_local[1] = fabs(moms_local[1*nc] - moms_target_local[1*nc]);
    }
    else {
      abs_diff_moms_local[1] = fabs(moms_local[1*nc] - moms_target_local[1*nc])/moms_target_local[1*nc];
    }
  }
}

void
gkyl_gyrokinetic_maxwellian_correct_all_moments_abs_diff_cu(const struct gkyl_range *conf_range, 
  int nc, const struct gkyl_array *moms_target, const struct gkyl_array *moms_iter, 
  struct gkyl_array *moms_abs_diff)
{
  int nblocks = conf_range->nblocks, nthreads = conf_range->nthreads;
  gkyl_gyrokinetic_maxwellian_correct_all_moments_abs_diff_cu_ker<<<nblocks, nthreads>>>(*conf_range, 
    nc, moms_target->on_dev, moms_iter->on_dev, moms_abs_diff->on_dev);
}

__global__ static void
gkyl_gyrokinetic_bimaxwellian_correct_all_moments_abs_diff_cu_ker(struct gkyl_range conf_range, 
  int nc, const struct gkyl_array *moms_target, const struct gkyl_array *moms_iter, 
  struct gkyl_array *abs_diff_moms)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *moms_local = (const double*) gkyl_array_cfetch(moms_iter, loc);
    const double *moms_target_local = (const double*) gkyl_array_cfetch(moms_target, loc);
    double *abs_diff_moms_local = (double*) gkyl_array_fetch(abs_diff_moms, loc);
    // Compute the absolute value of the difference of cell averages 
    // Note: max error found by follow-up thread-safe reduction operation
    // Also: for density, T_par/m, T_perp/m this error is a relative error compared to the target moment value
    // so that we can converge to the correct target moments in SI units and minimize finite precision issues.
    abs_diff_moms_local[0] = fabs(moms_local[0*nc] - moms_target_local[0*nc])/moms_target_local[0*nc];
    abs_diff_moms_local[2] = fabs(moms_local[2*nc] - moms_target_local[2*nc])/moms_target_local[2*nc];
    abs_diff_moms_local[3] = fabs(moms_local[3*nc] - moms_target_local[3*nc])/moms_target_local[3*nc];
    // However, u_par may be ~ 0 and if it is, we need to use absolute error. We can converge safely using
    // absolute error if u_par ~ O(1). Otherwise, we use relative error for u_par. 
    if (fabs(moms_target_local[1*nc]) < 1.0) {
      abs_diff_moms_local[1] = fabs(moms_local[1*nc] - moms_target_local[1*nc]);
    }
    else {
      abs_diff_moms_local[1] = fabs(moms_local[1*nc] - moms_target_local[1*nc])/moms_target_local[1*nc];
    }
  }
}

void
gkyl_gyrokinetic_bimaxwellian_correct_all_moments_abs_diff_cu(const struct gkyl_range *conf_range, 
  int nc, const struct gkyl_array *moms_target, const struct gkyl_array *moms_iter, 
  struct gkyl_array *moms_abs_diff)
{
  int nblocks = conf_range->nblocks, nthreads = conf_range->nthreads;
  gkyl_gyrokinetic_bimaxwellian_correct_all_moments_abs_diff_cu_ker<<<nblocks, nthreads>>>(*conf_range, 
    nc, moms_target->on_dev, moms_iter->on_dev, moms_abs_diff->on_dev);
}
