#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <assert.h>
#include <stdlib.h>
#include <string.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_prim_vars_gyrokinetic.h>
#include <gkyl_dg_prim_vars_gyrokinetic_priv.h>
#include <gkyl_util.h>
}

enum { upar, vtSq, prim, BAD };

static int
get_prim_id(const char *prim_nm)
{
  int prim_idx = BAD;

  if (strcmp(prim_nm, "upar") == 0) { // flow velocity
    prim_idx = upar;
  }
  else if (strcmp(prim_nm, "vtSq") == 0) { // vth^2
    prim_idx = vtSq;
  }
  else if (strcmp(prim_nm, "prim") == 0) { // combined (upar, vth^2)
    prim_idx = prim;
  }
  else {
    prim_idx = BAD;
  }    

  return prim_idx;
}

static int
v_num_prim(int vdim, int prim_id)
{
  int num_prim = 0;
  
  switch (prim_id) {
    case upar:
      num_prim = 1;
      break;

    case vtSq:
      num_prim = 1;
      break;

    case prim:
      num_prim = 2;
      break;    
      
    default: // can't happen
      break;
  }

  return num_prim;
}

__global__
static void
set_cu_ptrs(struct dg_prim_vars_type_gyrokinetic* pvt, int prim_id, enum gkyl_basis_type b_type, int vdim,
  int poly_order, int tblidx)
{
  const gkyl_dg_prim_vars_gyrokinetic_kern_list *dg_prim_vars_gyrokinetic_upar_kernels, 
    *dg_prim_vars_gyrokinetic_vtSq_kernels, *dg_prim_vars_gyrokinetic_kernels;

  // choose kernel tables based on basis-function type
  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      dg_prim_vars_gyrokinetic_upar_kernels = ser_dg_prim_vars_gyrokinetic_upar_kernels;
      dg_prim_vars_gyrokinetic_vtSq_kernels = ser_dg_prim_vars_gyrokinetic_vtSq_kernels;
      dg_prim_vars_gyrokinetic_kernels = ser_dg_prim_vars_gyrokinetic_kernels;
      break;

    default:
      assert(false);
      break;    
  }
  
  switch (prim_id) {
    case upar:
      pvt->pvt.kernel = dg_prim_vars_gyrokinetic_upar_kernels[tblidx].kernels[poly_order];
      break;

    case vtSq:
      pvt->pvt.kernel = dg_prim_vars_gyrokinetic_vtSq_kernels[tblidx].kernels[poly_order];
      break;

    case prim:
      pvt->pvt.kernel = dg_prim_vars_gyrokinetic_kernels[tblidx].kernels[poly_order];
      break;
      
    default: // can't happen
      break;
  }
}

struct gkyl_dg_prim_vars_type*
gkyl_dg_prim_vars_gyrokinetic_cu_dev_new(const struct gkyl_basis* cbasis,
  const struct gkyl_basis* pbasis, const char *prim_nm)
{
  assert(cbasis->poly_order == pbasis->poly_order);

  struct dg_prim_vars_type_gyrokinetic *pvt = (struct dg_prim_vars_type_gyrokinetic*)
    gkyl_malloc(sizeof(struct dg_prim_vars_type_gyrokinetic));
  
  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  pvt->pvt.cdim = cdim;
  pvt->pvt.vdim = vdim;
  pvt->pvt.poly_order = poly_order;
  pvt->pvt.num_config = cbasis->num_basis;

  int prim_id = get_prim_id(prim_nm);
  assert(prim_id != BAD);
  pvt->pvt.num_mom = v_num_prim(vdim, prim_id); // number of primitive variables

  pvt->pvt.flags = 0;
  GKYL_SET_CU_ALLOC(pvt->pvt.flags);
  pvt->pvt.ref_count = gkyl_ref_count_init(gkyl_dg_prim_vars_gyrokinetic_free);
  
  // copy struct to device
  struct dg_prim_vars_type_gyrokinetic *pvt_cu = (struct dg_prim_vars_type_gyrokinetic*)
    gkyl_cu_malloc(sizeof(struct dg_prim_vars_type_gyrokinetic));
  gkyl_cu_memcpy(pvt_cu, pvt, sizeof(struct dg_prim_vars_type_gyrokinetic), GKYL_CU_MEMCPY_H2D);

  assert(cv_index[cdim].vdim[vdim] != -1);

  set_cu_ptrs<<<1,1>>>(pvt_cu, prim_id, cbasis->b_type,
    vdim, poly_order, cv_index[cdim].vdim[vdim]);

  pvt->pvt.on_dev = &pvt_cu->pvt;
  
  return &pvt->pvt;
}
