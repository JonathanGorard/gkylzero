#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_calc_pkpm_vars.h>
#include <gkyl_dg_calc_pkpm_vars_priv.h>
#include <gkyl_util.h>
}

__global__ static void
gkyl_dg_calc_pkpm_vars_set_cu_kernel(gkyl_dg_calc_pkpm_vars* up,
  struct gkyl_nmat *As, struct gkyl_nmat *xs, struct gkyl_range conf_range,
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* p_ij, const struct gkyl_array* pkpm_div_ppar, 
  struct gkyl_array* cell_avg_prim)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*up->Ncomp;

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *p_ij_d = (const double*) gkyl_array_cfetch(p_ij, loc);
    const double *pkpm_div_ppar_d = (const double*) gkyl_array_cfetch(pkpm_div_ppar, loc);

    int* cell_avg_prim_d = (int*) gkyl_array_fetch(cell_avg_prim, loc);

    cell_avg_prim_d[0] = up->pkpm_set(count, As, xs, 
      vlasov_pkpm_moms_d, euler_pkpm_d, p_ij_d, pkpm_div_ppar_d);
  }
}

__global__ static void
gkyl_dg_calc_pkpm_vars_copy_cu_kernel(gkyl_dg_calc_pkpm_vars* up, 
  struct gkyl_nmat *xs, struct gkyl_range conf_range,
  struct gkyl_array* prim)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*up->Ncomp;

    double* prim_d = (double*) gkyl_array_fetch(prim, loc);

    up->pkpm_copy(count, xs, prim_d);
  }
}

// Host-side wrapper for pkpm primitive variable calculation
void gkyl_dg_calc_pkpm_vars_advance_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* p_ij, const struct gkyl_array* pkpm_div_ppar, 
  struct gkyl_array* cell_avg_prim, struct gkyl_array* prim)
{
  struct gkyl_range conf_range = up->mem_range;
  
  gkyl_dg_calc_pkpm_vars_set_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->As->on_dev, up->xs->on_dev, conf_range,
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, 
    p_ij->on_dev, pkpm_div_ppar->on_dev, cell_avg_prim->on_dev);

  if (up->poly_order > 1) {
    bool status = gkyl_nmat_linsolve_lu_pa(up->mem, up->As, up->xs);
    assert(status);
  }

  gkyl_dg_calc_pkpm_vars_copy_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->xs->on_dev, conf_range, prim->on_dev);
}

__global__ void
gkyl_calc_pkpm_vars_pressure_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* vlasov_pkpm_moms, struct gkyl_array* p_ij)
{ 
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *bvar_d = (const double*) gkyl_array_cfetch(bvar, loc);
    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);

    double *p_ij_d = (double*) gkyl_array_fetch(p_ij, loc);
    up->pkpm_pressure(bvar_d, vlasov_pkpm_moms_d, p_ij_d);
  }
}

// Host-side wrapper for pkpm pressure calculation
void gkyl_dg_calc_pkpm_vars_pressure_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* vlasov_pkpm_moms, struct gkyl_array* p_ij)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_calc_pkpm_vars_pressure_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    bvar->on_dev, vlasov_pkpm_moms->on_dev, p_ij->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_accel_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* prim, const struct gkyl_array* nu, 
  struct gkyl_array* pkpm_accel)
{
  int idxl[GKYL_MAX_DIM], idxc[GKYL_MAX_DIM], idxr[GKYL_MAX_DIM];
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idxc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long linc = gkyl_range_idx(&conf_range, idxc);

    const double *bvar_c = (const double*) gkyl_array_cfetch(bvar, linc);
    const double *prim_c = (const double*) gkyl_array_cfetch(prim, linc);

    // Only need nu in center cell
    const double *nu_d = (const double*) gkyl_array_cfetch(nu, linc);

    double *pkpm_accel_d = (double*) gkyl_array_fetch(pkpm_accel, linc);

    for (int dir=0; dir<up->cdim; ++dir) {
      gkyl_copy_int_arr(up->cdim, idxc, idxl);
      gkyl_copy_int_arr(up->cdim, idxc, idxr);

      idxl[dir] = idxl[dir]-1; idxr[dir] = idxr[dir]+1;

      long linl = gkyl_range_idx(&conf_range, idxl); 
      long linr = gkyl_range_idx(&conf_range, idxr);

      const double *bvar_l = (const double*) gkyl_array_cfetch(bvar, linl);
      const double *bvar_r = (const double*) gkyl_array_cfetch(bvar, linr);

      const double *prim_l = (const double*) gkyl_array_cfetch(prim, linl);
      const double *prim_r = (const double*) gkyl_array_cfetch(prim, linr);

      up->pkpm_accel[dir](up->conf_grid.dx, 
        bvar_l, bvar_c, bvar_r, 
        prim_l, prim_c, prim_r, 
        nu_d, pkpm_accel_d);
    }
  }
}

// Host-side wrapper for pkpm acceleration variable calculations with recovery or averaging
void
gkyl_dg_calc_pkpm_vars_accel_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* prim, const struct gkyl_array* nu, 
  struct gkyl_array* pkpm_accel)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_accel_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    bvar->on_dev, prim->on_dev, nu->on_dev, 
    pkpm_accel->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_integrated_vars_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* prim, struct gkyl_array* int_pkpm_vars)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *prim_d = (const double*) gkyl_array_cfetch(prim, loc);

    double *int_pkpm_vars_d = (double*) gkyl_array_fetch(int_pkpm_vars, loc);
    up->pkpm_int(vlasov_pkpm_moms_d, euler_pkpm_d, prim_d, int_pkpm_vars_d);
  }
}

// Host-side wrapper for pkpm integrated variables calculation
void
gkyl_dg_calc_pkpm_integrated_vars_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range,
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* prim, struct gkyl_array* int_pkpm_vars)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_integrated_vars_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, prim->on_dev, 
    int_pkpm_vars->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_source_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* qmem, const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm,
  struct gkyl_array* rhs)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *qmem_d = (const double*) gkyl_array_cfetch(qmem, loc);
    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);

    double *rhs_d = (double*) gkyl_array_fetch(rhs, loc);
    up->pkpm_source(qmem_d, vlasov_pkpm_moms_d, euler_pkpm_d, rhs_d);
  }
}

// Host-side wrapper for pkpm source term calculations
void
gkyl_dg_calc_pkpm_vars_source_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range,
  const struct gkyl_array* qmem, const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  struct gkyl_array* rhs)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_source_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    qmem->on_dev, vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, 
    rhs->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_io_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* vlasov_pkpm_moms, 
  const struct gkyl_array* euler_pkpm, const struct gkyl_array* p_ij, 
  const struct gkyl_array* prim, const struct gkyl_array* pkpm_accel, 
  struct gkyl_array* fluid_io, struct gkyl_array* pkpm_vars_io)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *p_ij_d = (const double*) gkyl_array_cfetch(p_ij, loc);
    const double *prim_d = (const double*) gkyl_array_cfetch(prim, loc);
    const double *pkpm_accel_d = (const double*) gkyl_array_cfetch(pkpm_accel, loc);

    double *fluid_io_d = (double*) gkyl_array_fetch(fluid_io, loc);
    double *pkpm_vars_io_d = (double*) gkyl_array_fetch(pkpm_vars_io, loc);
    up->pkpm_io(vlasov_pkpm_moms_d, euler_pkpm_d, p_ij_d, prim_d, pkpm_accel_d, 
      fluid_io_d, pkpm_vars_io_d);
  }
}

// Host-side wrapper for pkpm io. Computes conserved variables and copies primitive and acceleration variables to output array
void
gkyl_dg_calc_pkpm_vars_io_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_range *conf_range, const struct gkyl_array* vlasov_pkpm_moms, 
  const struct gkyl_array* euler_pkpm, const struct gkyl_array* p_ij, 
  const struct gkyl_array* prim, const struct gkyl_array* pkpm_accel, 
  struct gkyl_array* fluid_io, struct gkyl_array* pkpm_vars_io)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_io_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, p_ij->on_dev, prim->on_dev, pkpm_accel->on_dev, 
    fluid_io->on_dev, pkpm_vars_io->on_dev);
}

// CUDA kernel to set device pointers to pkpm vars kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_calc_pkpm_vars_set_cu_dev_ptrs(struct gkyl_dg_calc_pkpm_vars *up, enum gkyl_basis_type b_type,
  int cdim,int poly_order)
{
  up->pkpm_set = choose_pkpm_set_kern(b_type, cdim, poly_order);
  up->pkpm_copy = choose_pkpm_copy_kern(b_type, cdim, poly_order);
  up->pkpm_pressure = choose_pkpm_pressure_kern(b_type, cdim, poly_order);
  up->pkpm_source = choose_pkpm_source_kern(b_type, cdim, poly_order);
  up->pkpm_int = choose_pkpm_int_kern(b_type, cdim, poly_order);
  up->pkpm_io = choose_pkpm_io_kern(b_type, cdim, poly_order);
  // Fetch the kernels in each direction
  for (int d=0; d<cdim; ++d) 
    up->pkpm_accel[d] = choose_pkpm_accel_kern(d, b_type, cdim, poly_order);
}

gkyl_dg_calc_pkpm_vars*
gkyl_dg_calc_pkpm_vars_cu_dev_new(const struct gkyl_rect_grid *conf_grid, 
  const struct gkyl_basis* cbasis, const struct gkyl_range *mem_range)
{
  struct gkyl_dg_calc_pkpm_vars *up = (struct gkyl_dg_calc_pkpm_vars*) gkyl_malloc(sizeof(gkyl_dg_calc_pkpm_vars));

  up->conf_grid = *conf_grid;
  int nc = cbasis->num_basis;
  enum gkyl_basis_type b_type = cbasis->b_type;
  int cdim = cbasis->ndim;
  int poly_order = cbasis->poly_order;
  up->poly_order = poly_order;
  up->mem_range = *mem_range;
  up->Ncomp = 9;

  // There are Ncomp more linear systems to be solved 
  // 9 components: ux, uy, uz, 3*Txx/m, 3*Tyy/m, 3*Tzz/m, div(p_par b)/rho, p_perp/rho, rho/p_perp
  up->As = gkyl_nmat_cu_dev_new(up->Ncomp*mem_range->volume, nc, nc);
  up->xs = gkyl_nmat_cu_dev_new(up->Ncomp*mem_range->volume, nc, 1);
  up->mem = gkyl_nmat_linsolve_lu_cu_dev_new(up->As->num, up->As->nr);

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_calc_pkpm_vars *up_cu = (struct gkyl_dg_calc_pkpm_vars*) gkyl_cu_malloc(sizeof(gkyl_dg_calc_pkpm_vars));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_calc_pkpm_vars), GKYL_CU_MEMCPY_H2D);

  dg_calc_pkpm_vars_set_cu_dev_ptrs<<<1,1>>>(up_cu, b_type, cdim, poly_order);

  // set parent on_dev pointer
  up->on_dev = up_cu;
  
  return up;
}
