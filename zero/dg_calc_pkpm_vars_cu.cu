#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_calc_pkpm_vars.h>
#include <gkyl_dg_calc_pkpm_vars_priv.h>
#include <gkyl_util.h>
}

__global__ static void
gkyl_dg_calc_pkpm_vars_set_cu_kernel(gkyl_dg_calc_pkpm_vars* up,
  struct gkyl_nmat *As, struct gkyl_nmat *xs, struct gkyl_range conf_range,
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* pkpm_div_ppar, struct gkyl_array* cell_avg_prim)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*up->Ncomp;

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *pkpm_div_ppar_d = (const double*) gkyl_array_cfetch(pkpm_div_ppar, loc);

    int* cell_avg_prim_d = (int*) gkyl_array_fetch(cell_avg_prim, loc);

    cell_avg_prim_d[0] = up->pkpm_set(count, As, xs, 
      vlasov_pkpm_moms_d, euler_pkpm_d, pkpm_div_ppar_d);
  }
}

__global__ static void
gkyl_dg_calc_pkpm_vars_copy_cu_kernel(gkyl_dg_calc_pkpm_vars* up, 
  struct gkyl_nmat *xs, struct gkyl_range conf_range,
  struct gkyl_array* prim)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp systems in each cell)
    long count = linc1*up->Ncomp;

    double* prim_d = (double*) gkyl_array_fetch(prim, loc);

    up->pkpm_copy(count, xs, prim_d);
  }
}

// Host-side wrapper for pkpm primitive variable calculation
void gkyl_dg_calc_pkpm_vars_advance_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* pkpm_div_ppar, struct gkyl_array* cell_avg_prim, 
  struct gkyl_array* prim)
{
  struct gkyl_range conf_range = up->mem_range;
  
  gkyl_dg_calc_pkpm_vars_set_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->As->on_dev, up->xs->on_dev, conf_range,
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, 
    p_ij->on_dev, pkpm_div_ppar->on_dev, cell_avg_prim->on_dev);

  if (up->poly_order > 1) {
    bool status = gkyl_nmat_linsolve_lu_pa(up->mem, up->As, up->xs);
    assert(status);
  }

  gkyl_dg_calc_pkpm_vars_copy_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->xs->on_dev, conf_range, prim->on_dev);
}

__global__ static void
gkyl_dg_calc_pkpm_vars_surf_set_cu_kernel(gkyl_dg_calc_pkpm_vars* up,
  struct gkyl_nmat *As, struct gkyl_nmat *xs, struct gkyl_range conf_range,
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* p_ij, const struct gkyl_array* cell_avg_prim)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp_surf systems in each cell)
    long count = linc1*up->Ncomp_surf;

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *p_ij_d = (const double*) gkyl_array_cfetch(p_ij, loc);
    const int* cell_avg_prim_d = (const int*) gkyl_array_cfetch(cell_avg_prim, loc);

    up->pkpm_surf_set(count, As, xs, 
      vlasov_pkpm_moms_d, euler_pkpm_d, p_ij_d, cell_avg_prim_d);
  }
}

__global__ static void
gkyl_dg_calc_pkpm_vars_surf_copy_cu_kernel(gkyl_dg_calc_pkpm_vars* up, 
  struct gkyl_nmat *xs, struct gkyl_range conf_range,
  struct gkyl_array* prim_surf)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);
    // fetch the correct count in the matrix (since we solve Ncomp_surf systems in each cell)
    long count = linc1*up->Ncomp_surf;

    double* prim_surf_d = (double*) gkyl_array_fetch(prim_surf, loc);

    up->pkpm_surf_copy(count, xs, prim_surf_d);
  }
}

// Host-side wrapper for pkpm surface primitive variable calculation
void gkyl_dg_calc_pkpm_vars_surf_advance_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* p_ij, const struct gkyl_array* cell_avg_prim, 
  struct gkyl_array* prim_surf)
{
  struct gkyl_range conf_range = up->mem_range;
  
  gkyl_dg_calc_pkpm_vars_surf_set_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->As_surf->on_dev, up->xs_surf->on_dev, conf_range,
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, 
    p_ij->on_dev, cell_avg_prim->on_dev);

  if (up->poly_order > 1 && up->cdim > 1) {
    bool status = gkyl_nmat_linsolve_lu_pa(up->mem_surf, up->As_surf, up->xs_surf);
    assert(status);
  }

  gkyl_dg_calc_pkpm_vars_surf_copy_cu_kernel<<<conf_range.nblocks, conf_range.nthreads>>>(up->on_dev,
    up->xs_surf->on_dev, conf_range, prim_surf->on_dev);
}

__global__ void
gkyl_calc_pkpm_vars_pressure_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* vlasov_pkpm_moms, struct gkyl_array* p_ij)
{ 
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *bvar_d = (const double*) gkyl_array_cfetch(bvar, loc);
    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);

    double *p_ij_d = (double*) gkyl_array_fetch(p_ij, loc);
    up->pkpm_pressure(bvar_d, vlasov_pkpm_moms_d, p_ij_d);
  }
}

// Host-side wrapper for pkpm pressure calculation
void gkyl_dg_calc_pkpm_vars_pressure_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* vlasov_pkpm_moms, struct gkyl_array* p_ij)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_calc_pkpm_vars_pressure_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    bvar->on_dev, vlasov_pkpm_moms->on_dev, p_ij->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_accel_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* prim_surf, 
  const struct gkyl_array* prim, const struct gkyl_array* nu, 
  struct gkyl_array* pkpm_lax, struct gkyl_array* pkpm_accel)
{
  int cdim = up->cdim;
  int idxl[GKYL_MAX_DIM], idxc[GKYL_MAX_DIM], idxr[GKYL_MAX_DIM];
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idxc);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long linc = gkyl_range_idx(&conf_range, idxc);

    const double *bvar_c = (const double*) gkyl_array_cfetch(bvar, linc);
    const double *prim_surf_c = (const double*) gkyl_array_cfetch(prim_surf, linc);
    // Only need nu and the volume expansion of the primitive moments in center cell
    const double *prim_d = (const double*) gkyl_array_cfetch(prim, linc);
    const double *nu_d = (const double*) gkyl_array_cfetch(nu, linc);

    double *pkpm_lax_d = (double*) gkyl_array_fetch(pkpm_lax, linc);
    double *pkpm_accel_d = (double*) gkyl_array_fetch(pkpm_accel, linc);

    for (int dir=0; dir<cdim; ++dir) {
      gkyl_copy_int_arr(cdim, idxc, idxl);
      gkyl_copy_int_arr(cdim, idxc, idxr);

      idxl[dir] = idxl[dir]-1; idxr[dir] = idxr[dir]+1;

      long linl = gkyl_range_idx(&conf_range, idxl); 
      long linr = gkyl_range_idx(&conf_range, idxr);

      const double *bvar_l = (const double*) gkyl_array_cfetch(bvar, linl);
      const double *bvar_r = (const double*) gkyl_array_cfetch(bvar, linr);

      const double *prim_surf_l = (const double*) gkyl_array_cfetch(prim_surf, linl);
      const double *prim_surf_r = (const double*) gkyl_array_cfetch(prim_surf, linr);
      
      up->pkpm_accel[dir](up->conf_grid.dx, 
        bvar_l, bvar_c, bvar_r, 
        prim_surf_l, prim_surf_c, prim_surf_r, 
        prim_d, nu_d,
        pkpm_lax_d, pkpm_accel_d);
    }
  }
}

// Host-side wrapper for pkpm acceleration variable calculations with recovery or averaging
void
gkyl_dg_calc_pkpm_vars_accel_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range, 
  const struct gkyl_array* bvar, const struct gkyl_array* prim_surf, 
  const struct gkyl_array* prim, const struct gkyl_array* nu, 
  struct gkyl_array* pkpm_lax, struct gkyl_array* pkpm_accel)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_accel_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    bvar->on_dev, prim_surf->on_dev, prim->on_dev, nu->on_dev, 
    pkpm_lax->on_dev, pkpm_accel->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_integrated_vars_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* prim, struct gkyl_array* int_pkpm_vars)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *prim_d = (const double*) gkyl_array_cfetch(prim, loc);

    double *int_pkpm_vars_d = (double*) gkyl_array_fetch(int_pkpm_vars, loc);
    up->pkpm_int(vlasov_pkpm_moms_d, euler_pkpm_d, prim_d, int_pkpm_vars_d);
  }
}

// Host-side wrapper for pkpm integrated variables calculation
void
gkyl_dg_calc_pkpm_integrated_vars_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range,
  const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  const struct gkyl_array* prim, struct gkyl_array* int_pkpm_vars)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_integrated_vars_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, prim->on_dev, 
    int_pkpm_vars->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_source_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* qmem, const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm,
  struct gkyl_array* rhs)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *qmem_d = (const double*) gkyl_array_cfetch(qmem, loc);
    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);

    double *rhs_d = (double*) gkyl_array_fetch(rhs, loc);
    up->pkpm_source(qmem_d, vlasov_pkpm_moms_d, euler_pkpm_d, rhs_d);
  }
}

// Host-side wrapper for pkpm source term calculations
void
gkyl_dg_calc_pkpm_vars_source_cu(struct gkyl_dg_calc_pkpm_vars *up, const struct gkyl_range *conf_range,
  const struct gkyl_array* qmem, const struct gkyl_array* vlasov_pkpm_moms, const struct gkyl_array* euler_pkpm, 
  struct gkyl_array* rhs)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_source_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    qmem->on_dev, vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, 
    rhs->on_dev);
}

__global__ void
gkyl_dg_calc_pkpm_vars_io_cu_kernel(struct gkyl_dg_calc_pkpm_vars *up, struct gkyl_range conf_range, 
  const struct gkyl_array* vlasov_pkpm_moms, 
  const struct gkyl_array* euler_pkpm, const struct gkyl_array* p_ij, 
  const struct gkyl_array* prim, const struct gkyl_array* pkpm_accel, 
  struct gkyl_array* fluid_io, struct gkyl_array* pkpm_vars_io)
{
  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&conf_range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long loc = gkyl_range_idx(&conf_range, idx);

    const double *vlasov_pkpm_moms_d = (const double*) gkyl_array_cfetch(vlasov_pkpm_moms, loc);
    const double *euler_pkpm_d = (const double*) gkyl_array_cfetch(euler_pkpm, loc);
    const double *p_ij_d = (const double*) gkyl_array_cfetch(p_ij, loc);
    const double *prim_d = (const double*) gkyl_array_cfetch(prim, loc);
    const double *pkpm_accel_d = (const double*) gkyl_array_cfetch(pkpm_accel, loc);

    double *fluid_io_d = (double*) gkyl_array_fetch(fluid_io, loc);
    double *pkpm_vars_io_d = (double*) gkyl_array_fetch(pkpm_vars_io, loc);
    up->pkpm_io(vlasov_pkpm_moms_d, euler_pkpm_d, p_ij_d, prim_d, pkpm_accel_d, 
      fluid_io_d, pkpm_vars_io_d);
  }
}

// Host-side wrapper for pkpm io. Computes conserved variables and copies primitive and acceleration variables to output array
void
gkyl_dg_calc_pkpm_vars_io_cu(struct gkyl_dg_calc_pkpm_vars *up, 
  const struct gkyl_range *conf_range, const struct gkyl_array* vlasov_pkpm_moms, 
  const struct gkyl_array* euler_pkpm, const struct gkyl_array* p_ij, 
  const struct gkyl_array* prim, const struct gkyl_array* pkpm_accel, 
  struct gkyl_array* fluid_io, struct gkyl_array* pkpm_vars_io)
{
  int nblocks = conf_range->nblocks;
  int nthreads = conf_range->nthreads;
  gkyl_dg_calc_pkpm_vars_io_cu_kernel<<<nblocks, nthreads>>>(up->on_dev, *conf_range, 
    vlasov_pkpm_moms->on_dev, euler_pkpm->on_dev, p_ij->on_dev, prim->on_dev, pkpm_accel->on_dev, 
    fluid_io->on_dev, pkpm_vars_io->on_dev);
}

// CUDA kernel to set device pointers to pkpm vars kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
dg_calc_pkpm_vars_set_cu_dev_ptrs(struct gkyl_dg_calc_pkpm_vars *up, enum gkyl_basis_type b_type,
  int cdim,int poly_order)
{
  up->pkpm_set = choose_pkpm_set_kern(b_type, cdim, poly_order);
  up->pkpm_surf_set = choose_pkpm_surf_set_kern(b_type, cdim, poly_order);
  up->pkpm_copy = choose_pkpm_copy_kern(b_type, cdim, poly_order);
  up->pkpm_surf_copy = choose_pkpm_surf_copy_kern(b_type, cdim, poly_order);
  up->pkpm_pressure = choose_pkpm_pressure_kern(b_type, cdim, poly_order);
  up->pkpm_source = choose_pkpm_source_kern(b_type, cdim, poly_order);
  up->pkpm_int = choose_pkpm_int_kern(b_type, cdim, poly_order);
  up->pkpm_io = choose_pkpm_io_kern(b_type, cdim, poly_order);
  // Fetch the kernels in each direction
  for (int d=0; d<cdim; ++d) 
    up->pkpm_accel[d] = choose_pkpm_accel_kern(d, b_type, cdim, poly_order);
}

gkyl_dg_calc_pkpm_vars*
gkyl_dg_calc_pkpm_vars_cu_dev_new(const struct gkyl_rect_grid *conf_grid, 
  const struct gkyl_basis* cbasis, const struct gkyl_range *mem_range)
{
  struct gkyl_dg_calc_pkpm_vars *up = (struct gkyl_dg_calc_pkpm_vars*) gkyl_malloc(sizeof(gkyl_dg_calc_pkpm_vars));

  up->conf_grid = *conf_grid;
  int nc = cbasis->num_basis;
  int cdim = cbasis->ndim;
  int poly_order = cbasis->poly_order;
  int nc_surf = cbasis->num_basis/(poly_order+1); // *only valid for tensor bases for cdim > 1*
  enum gkyl_basis_type b_type = cbasis->b_type;
  up->cdim = cdim;
  up->poly_order = poly_order;
  up->Ncomp = 6;
  up->Ncomp_surf = 2*cdim*3+2*cdim;

  // There are Ncomp*range->volume linear systems to be solved 
  // 6 components: ux, uy, uz, div(p_par b)/rho, p_perp/rho, rho/p_perp
  up->As = gkyl_nmat_cu_dev_new(up->Ncomp*mem_range->volume, nc, nc);
  up->xs = gkyl_nmat_cu_dev_new(up->Ncomp*mem_range->volume, nc, 1);
  up->mem = gkyl_nmat_linsolve_lu_cu_dev_new(up->As->num, up->As->nr);

  // There are Ncomp_surf*range->volume linear systems to be solved 
  // Each linear system is nc_surf x nc_surf (only solved over the surface basis and only when poly_order and cdim > 1)
  // 2*cdim*3+2*cdim components: ux, uy, uz (3 components) at the left and right of the cell (2 components) in each dimension (cdim components)
  // Also solves for 3*Txx/m at the left and right x surfaces, 3*Tyy/m at the left and right y surfaces, and 3*Tzz/m at the left and right y surfaces
  up->As_surf = gkyl_nmat_new(up->Ncomp_surf*mem_range->volume, nc_surf, nc_surf);
  up->xs_surf = gkyl_nmat_new(up->Ncomp_surf*mem_range->volume, nc_surf, 1);
  up->mem_surf = gkyl_nmat_linsolve_lu_new(up->As_surf->num, up->As_surf->nr);

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  struct gkyl_dg_calc_pkpm_vars *up_cu = (struct gkyl_dg_calc_pkpm_vars*) gkyl_cu_malloc(sizeof(gkyl_dg_calc_pkpm_vars));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_dg_calc_pkpm_vars), GKYL_CU_MEMCPY_H2D);

  dg_calc_pkpm_vars_set_cu_dev_ptrs<<<1,1>>>(up_cu, b_type, cdim, poly_order);

  // set parent on_dev pointer
  up->on_dev = up_cu;
  
  return up;
}
