#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_wv_maxwell.h>    
#include <gkyl_wv_maxwell_priv.h>
}

#include <cassert>

// CUDA kernel to set device pointers to wv_eqn maxwell kernel functions
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void 
wv_maxwell_set_cu_dev_ptrs(struct wv_maxwell *maxwell)
{
  maxwell->eqn.waves_func = wave_roe;
  maxwell->eqn.qfluct_func = qfluct_roe;
  maxwell->eqn.max_speed_func = max_speed;
  maxwell->eqn.rotate_to_local_func = rot_to_local;
  maxwell->eqn.rotate_to_global_func = rot_to_global;

  maxwell->eqn.wall_bc_func = maxwell_wall;
}

struct gkyl_wv_eqn*
gkyl_wv_maxwell_cu_dev_new(double c, double e_fact, double b_fact)
{
  struct wv_maxwell *maxwell = gkyl_malloc(sizeof(struct wv_maxwell));

  maxwell->eqn.type = GKYL_EQN_MAXWELL;
  maxwell->eqn.num_equations = 8;
  maxwell->eqn.num_waves = 6;
  
  maxwell->c = c;
  maxwell->e_fact = e_fact;
  maxwell->b_fact = b_fact;

  maxwell->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(maxwell->eqn.flags);
  maxwell->eqn.ref_count = gkyl_ref_count_init(gkyl_maxwell_free);

  // copy the host struct to device struct
  struct wv_maxwell *maxwell_cu = (struct wv_maxwell*) gkyl_cu_malloc(sizeof(struct wv_maxwell));
  gkyl_cu_memcpy(maxwell_cu, maxwell, sizeof(struct wv_maxwell), GKYL_CU_MEMCPY_H2D);

  wv_maxwell_set_cu_dev_ptrs<<<1,1>>>(maxwell_cu);

  maxwell->eqn.on_dev = &maxwell_cu->eqn; // CPU eqn obj points to itself
  return &maxwell->eqn;
}