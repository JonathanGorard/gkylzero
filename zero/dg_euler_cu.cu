#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_euler.h>    
#include <gkyl_dg_euler_priv.h>
}

#include <cassert>

#define CK(lst,cdim,poly_order) lst[cdim-1].kernels[poly_order]

// CUDA kernel to set pointer to auxiliary fields.
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_euler_set_auxfields_cu_kernel(const struct gkyl_dg_eqn *eqn, const struct gkyl_array *u_i, const struct gkyl_array *p_ij)
{
  struct dg_euler *euler = container_of(eqn, struct dg_euler, eqn);
  euler->auxfields.u_i = uvar;
  euler->auxfields.p_ij = pvar;
}

// Host-side wrapper for set_auxfields_cu_kernel
void
gkyl_euler_set_auxfields_cu(const struct gkyl_dg_eqn *eqn, struct gkyl_dg_euler_auxfields auxin)
{
  gkyl_euler_set_auxfields_cu_kernel<<<1,1>>>(eqn, auxin.u_i->on_dev, auxin.p_ij->on_dev);
}

__global__ void static
dg_euler_set_cu_dev_ptrs(struct dg_euler* euler, enum gkyl_basis_type b_type, int cdim, int poly_order)
{
  euler->auxfields.u_i = 0; 
  euler->auxfields.p_ij = 0; 

  const gkyl_dg_euler_vol_kern_list *vol_kernels;
  const gkyl_dg_euler_surf_kern_list *surf_x_kernels, *surf_y_kernels, *surf_z_kernels;  
  
  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      vol_kernels = ser_vol_kernels;
      surf_x_kernels = ser_surf_x_kernels;
      surf_y_kernels = ser_surf_y_kernels;
      surf_z_kernels = ser_surf_z_kernels;
      break;

    default:
      assert(false);
      break;    
  }  
  
  euler->eqn.vol_term = vol;
  euler->eqn.surf_term = surf;
  euler->eqn.boundary_surf_term = boundary_surf;

  euler->vol =  CK(vol_kernels, cdim, poly_order);

  euler->surf[0] = CK(surf_x_kernels, cdim, poly_order);
  if (cdim>1)
    euler->surf[1] = CK(surf_y_kernels, cdim, poly_order);
  if (cdim>2)
    euler->surf[2] = CK(surf_z_kernels, cdim, poly_order);
}

struct gkyl_dg_eqn*
gkyl_dg_euler_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_range* conf_range, 
  double gas_gamma)
{
  struct dg_euler *euler = (struct dg_euler*) gkyl_malloc(sizeof(struct dg_euler));

  // set basic parameters
  euler->eqn.num_equations = 5;
  euler->gas_gamma = gas_gamma;

  euler->conf_range = *conf_range;

  euler->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(euler->eqn.flags);
  euler->eqn.ref_count = gkyl_ref_count_init(gkyl_euler_free);

  // copy the host struct to device struct
  struct dg_euler *euler_cu = (struct dg_euler*) gkyl_cu_malloc(sizeof(struct dg_euler));
  gkyl_cu_memcpy(euler_cu, euler, sizeof(struct dg_euler), GKYL_CU_MEMCPY_H2D);
  dg_euler_set_cu_dev_ptrs<<<1,1>>>(euler_cu, cbasis->b_type, cbasis->ndim, cbasis->poly_order);

  // set parent on_dev pointer
  euler->eqn.on_dev = &euler_cu->eqn;

  return &euler->eqn;
}
