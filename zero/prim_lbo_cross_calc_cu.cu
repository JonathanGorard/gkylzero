#include "hip/hip_runtime.h"
/* -*- c++ -*- */
extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_mat.h>
#include <gkyl_prim_lbo_cross_calc.h>
#include <gkyl_prim_lbo_cross_calc_priv.h>
#include <gkyl_prim_lbo_vlasov_priv.h>
#include <gkyl_util.h>
}

static void
gkyl_get_array_range_kernel_launch_dims(dim3* dimGrid, dim3* dimBlock, gkyl_range range)
{
  int volume = range.volume;
  int ndim = range.ndim;
  // ac1 = size of last dimension of range (fastest moving dimension)
  int ac1 = range.iac[ndim-1] > 0 ? range.iac[ndim-1] : 1;
  dimBlock->x = min(ac1, GKYL_DEFAULT_NUM_THREADS);
  dimGrid->x = gkyl_int_div_up(ac1, dimBlock->x);

  dimBlock->y = gkyl_int_div_up(GKYL_DEFAULT_NUM_THREADS, ac1);
  dimGrid->y = gkyl_int_div_up(volume, ac1*dimBlock->y);
}

__global__ static void
gkyl_prim_lbo_cross_calc_set_cu_ker(gkyl_prim_lbo_cross_calc* calc,
  struct gkyl_nmat *As, struct gkyl_nmat *xs,
  struct gkyl_basis cbasis, struct gkyl_range conf_rng,
  int nspecies, const double betaGreenep1, const double self_m,
  const struct gkyl_array* self_u, const struct gkyl_array* self_vtsq,
  const double cross_m[GKYL_MAX_SPECIES], struct gkyl_array* cross_u[GKYL_MAX_SPECIES],
  struct gkyl_array* cross_vtsq[GKYL_MAX_SPECIES],
  const struct gkyl_array* moms, const struct gkyl_array* boundary_corrections)
{
  int ndim = cbasis.ndim;
  long count = 0;
  int idx[GKYL_MAX_DIM];

  // 2D thread grid
  // linc1 = c + n*idx1 (contiguous data, including component index c, with idx1 = 0,.., ac1-1)
  // long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
  // linc2 = idx2 + ac2*idx3 + ...
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_rng.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // full linear cell index (not including components) is 
    // idx1 + ac1*idx2 + ac1*ac2*idx3 + ... = idx1 + ac1*linc2.
    // we want to find the start linear index of each contiguous data block, 
    // which corresponds to idx1 = 0. 
    // so linear index of start of contiguous block is ac1*linc2.
    //if (linc1 < ac1) {
      gkyl_sub_range_inv_idx(&conf_rng, linc1, idx);
      long start = gkyl_range_idx(&conf_rng, idx);

      for (int n=1; n<nspecies; ++n) {
        struct gkyl_mat lhs = gkyl_nmat_get(As, count);
        struct gkyl_mat rhs = gkyl_nmat_get(xs, count);
	const double *self_u_d = (const double*) gkyl_array_cfetch(self_u, start);
	const double *self_vtsq_d = (const double*) gkyl_array_cfetch(self_vtsq, start);
	const double *cross_u_d = (const double*) gkyl_array_fetch(cross_u[n], start);
	const double *cross_vtsq_d = (const double*) gkyl_array_fetch(cross_vtsq[n], start);
        const double *moms_d = (const double*) gkyl_array_cfetch(moms, start);
        const double *boundary_corrections_d = (const double*) gkyl_array_cfetch(boundary_corrections, start);

        gkyl_mat_clear(&lhs, 0.0); gkyl_mat_clear(&rhs, 0.0);

        calc->prim->cross_prim(calc->prim, &lhs, &rhs, betaGreenep1, self_m,
	  self_u_d, self_vtsq_d,
	  cross_m[n], cross_u_d, cross_vtsq_d, moms_d,
	  boundary_corrections_d
	);
	count += 1;
	//}
    }
  }
}

__global__ static void
gkyl_prim_lbo_copy_sol_cu_ker(struct gkyl_nmat *xs,
  struct gkyl_basis cbasis, struct gkyl_range conf_rng,
  int nc, int vdim, int nspecies,
  struct gkyl_array* u_out[GKYL_MAX_DIM], struct gkyl_array* vtsq_out[GKYL_MAX_DIM])
{
  int ndim = cbasis.ndim;
  long count = 0;
  int idx[GKYL_MAX_DIM];

  // ac1 = size of last dimension of range (fastest moving dimension)
  long ac1 = conf_rng.iac[ndim-1] > 0 ? conf_rng.iac[ndim-1] : 1;
  
  // 2D thread grid
  // linc1 = c + n*idx1 (contiguous data, including component index c, with idx1 = 0,.., ac1-1)
  //long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
  // linc2 = idx2 + ac2*idx3 + ...
  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < conf_rng.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // full linear cell index (not including components) is 
    // idx1 + ac1*idx2 + ac1*ac2*idx3 + ... = idx1 + ac1*linc2.
    // we want to find the start linear index of each contiguous data block, 
    // which corresponds to idx1 = 0. 
    // so linear index of start of contiguous block is ac1*linc2.
    //if (linc1 < ac1) {
      gkyl_sub_range_inv_idx(&conf_rng, linc1, idx);
      long start = gkyl_range_idx(&conf_rng, idx);

      for (int n=0; n<nspecies; ++n) {
        struct gkyl_mat out_d = gkyl_nmat_get(xs, count);
        double *u_d = (double*) gkyl_array_fetch(u_out[n], start);
        double *vtsq_d = (double*) gkyl_array_fetch(vtsq_out[n], start);
    
        prim_lbo_copy_sol(&out_d, nc, vdim, u_d, vtsq_d);
      }
      //}
  }
}

void
gkyl_prim_lbo_cross_calc_advance_cu(gkyl_prim_lbo_cross_calc* calc, struct gkyl_basis cbasis,
  struct gkyl_range conf_rng, const double betaGreenep1, const double self_m,
  const struct gkyl_array* self_u, const struct gkyl_array* self_vtsq,
  const double cross_m[GKYL_MAX_SPECIES], struct gkyl_array* cross_u[GKYL_MAX_SPECIES],
  struct gkyl_array* cross_vtsq[GKYL_MAX_SPECIES], const struct gkyl_array* moms,
  const struct gkyl_array* boundary_corrections, struct gkyl_array* u_out[GKYL_MAX_SPECIES],
  struct gkyl_array* vtsq_out[GKYL_MAX_SPECIES])
{
  int nspecies = calc->nspecies;

  struct gkyl_array *cross_us[GKYL_MAX_SPECIES];
  struct gkyl_array *cross_vtsqs[GKYL_MAX_SPECIES];
  struct gkyl_array *u_outs[GKYL_MAX_SPECIES];
  struct gkyl_array *vtsq_outs[GKYL_MAX_SPECIES];
  
  for (int n=0; n<nspecies; ++n) {
    cross_us[n] = cross_u[n]->on_dev;
    cross_vtsqs[n] = cross_vtsq[n]->on_dev;
  }

  int nc = cbasis.num_basis;
  int vdim = calc->prim->pdim - calc->prim->cdim;
  int N = nc*(vdim + 1);
  
  if (calc->is_first) {
    calc->As = gkyl_nmat_cu_dev_new(nspecies*conf_rng.volume, N, N);
    calc->xs = gkyl_nmat_cu_dev_new(nspecies*conf_rng.volume, N, 1);
    calc->mem = gkyl_nmat_linsolve_lu_cu_dev_new(calc->As->num, calc->As->nr);
    calc->is_first = false;
  }

  gkyl_prim_lbo_cross_calc_set_cu_ker<<<conf_rng.nblocks, conf_rng.nthreads>>>(calc->on_dev,
    calc->As->on_dev, calc->xs->on_dev, cbasis, conf_rng,
    nspecies, betaGreenep1, self_m, self_u->on_dev, self_vtsq->on_dev,
    cross_m, cross_us, cross_vtsqs,
    moms->on_dev, boundary_corrections->on_dev);
  
  bool status = gkyl_nmat_linsolve_lu_pa(calc->mem, calc->As, calc->xs);

  gkyl_prim_lbo_copy_sol_cu_ker<<<conf_rng.nblocks, conf_rng.nthreads>>>(calc->xs->on_dev,
    cbasis, conf_rng, nc, vdim, nspecies,
    u_outs, vtsq_outs);
}

gkyl_prim_lbo_cross_calc*
gkyl_prim_lbo_cross_calc_cu_dev_new(const struct gkyl_rect_grid *grid,
  struct gkyl_prim_lbo_type *prim, int nspecies)
{
  gkyl_prim_lbo_cross_calc *up = (gkyl_prim_lbo_cross_calc*) gkyl_malloc(sizeof(gkyl_prim_lbo_cross_calc));
  up->grid = *grid;
  up->prim = prim;
  up->nspecies = nspecies;

  up->is_first = true;
  up->As = up->xs = 0;
  up->mem = 0;

  struct gkyl_prim_lbo_type *pt = gkyl_prim_lbo_type_acquire(prim);
  up->prim = pt->on_dev; // so memcpy below gets dev copy

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  gkyl_prim_lbo_cross_calc *up_cu = (gkyl_prim_lbo_cross_calc*) gkyl_cu_malloc(sizeof(gkyl_prim_lbo_cross_calc));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_prim_lbo_cross_calc), GKYL_CU_MEMCPY_H2D);

  up->prim = pt; // host portion of struct should have host copy
  up->on_dev = up_cu; // host pointer
  
  return up;
}
