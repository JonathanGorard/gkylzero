#include "hip/hip_runtime.h"
/* -*- c++ -*- */
extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_mat.h>
#include <gkyl_prim_lbo_cross_calc.h>
#include <gkyl_prim_lbo_cross_calc_priv.h>
#include <gkyl_prim_lbo_vlasov_priv.h>
#include <gkyl_util.h>
#include <stdio.h>
}

static void
gkyl_get_array_range_kernel_launch_dims(dim3* dimGrid, dim3* dimBlock, gkyl_range range)
{
  int volume = range.volume;
  int ndim = range.ndim;
  // ac1 = size of last dimension of range (fastest moving dimension)
  int ac1 = range.iac[ndim-1] > 0 ? range.iac[ndim-1] : 1;
  dimBlock->x = min(ac1, GKYL_DEFAULT_NUM_THREADS);
  dimGrid->x = gkyl_int_div_up(ac1, dimBlock->x);

  dimBlock->y = gkyl_int_div_up(GKYL_DEFAULT_NUM_THREADS, ac1);
  dimGrid->y = gkyl_int_div_up(volume, ac1*dimBlock->y);
}

__global__ static void
gkyl_prim_lbo_cross_calc_set_cu_ker(gkyl_prim_lbo_cross_calc* calc,
  struct gkyl_nmat *As, struct gkyl_nmat *xs,
  struct gkyl_basis cbasis, const struct gkyl_range conf_rng,
  const struct gkyl_array *greene,
  double self_m, const struct gkyl_array *self_u, const struct gkyl_array *self_vtsq,
  double cross_m, const struct gkyl_array *cross_u, const struct gkyl_array *cross_vtsq, 
  const struct gkyl_array *moms, const struct gkyl_array *boundary_corrections)
{
  int ndim = cbasis.ndim;
  int idx[GKYL_MAX_DIM];
  // ac1 = size of last dimension of range (fastest moving dimension)
  long ac1 = conf_rng.iac[ndim-1] > 0 ? conf_rng.iac[ndim-1] : 1;

  // 2D thread grid
  // linc1 = c + n*idx1 (contiguous data, including component index c, with idx1 = 0,.., ac1-1)
  long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
  // linc2 = idx2 + ac2*idx3 + ...
  for (unsigned long linc2 = threadIdx.y + blockIdx.y*blockDim.y;
      linc2 < conf_rng.volume/ac1;
      linc2 += gridDim.y*blockDim.y)
  {
    // full linear cell index (not including components) is 
    // idx1 + ac1*idx2 + ac1*ac2*idx3 + ... = idx1 + ac1*linc2.
    // we want to find the start linear index of each contiguous data block, 
    // which corresponds to idx1 = 0. 
    // so linear index of start of contiguous block is ac1*linc2.
    if (linc1 < ac1) {
      gkyl_sub_range_inv_idx(&conf_rng, linc1 + ac1*linc2, idx);
      long start = gkyl_range_idx(&conf_rng, idx);
    
      struct gkyl_mat lhs = gkyl_nmat_get(As, linc1 + ac1*linc2);
      struct gkyl_mat rhs = gkyl_nmat_get(xs, linc1 + ac1*linc2);
      const double *self_u_d = (const double*) gkyl_array_cfetch(self_u, start);
      const double *greene_d = (const double*) gkyl_array_cfetch(greene, start);
      const double *self_vtsq_d = (const double*) gkyl_array_cfetch(self_vtsq, start);
      const double *cross_u_d = (const double*) gkyl_array_cfetch(cross_u, start);
      const double *cross_vtsq_d = (const double*) gkyl_array_cfetch(cross_vtsq, start);
      const double *moms_d = (const double*) gkyl_array_cfetch(moms, start);
      const double *boundary_corrections_d = (const double*) gkyl_array_cfetch(boundary_corrections, start);
      
      gkyl_mat_clear(&lhs, 0.0); gkyl_mat_clear(&rhs, 0.0);

      calc->prim->cross_prim(calc->prim, &lhs, &rhs, idx, greene_d, 
        self_m, self_u_d, self_vtsq_d,
        cross_m, cross_u_d, cross_vtsq_d, 
        moms_d, boundary_corrections_d
      );
    }
  }
}

__global__ static void
gkyl_prim_lbo_copy_sol_cu_ker(struct gkyl_nmat *xs,
  struct gkyl_basis cbasis, struct gkyl_range conf_rng,
  int nc, int vdim, 
  struct gkyl_array* u_out, struct gkyl_array* vtsq_out)
{
  int ndim = cbasis.ndim;
  int idx[GKYL_MAX_DIM];

  // ac1 = size of last dimension of range (fastest moving dimension)
  long ac1 = conf_rng.iac[ndim-1] > 0 ? conf_rng.iac[ndim-1] : 1;
  
  // 2D thread grid
  // linc1 = c + n*idx1 (contiguous data, including component index c, with idx1 = 0,.., ac1-1)
  long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
  // linc2 = idx2 + ac2*idx3 + ...
  for (unsigned long linc2 = threadIdx.y + blockIdx.y*blockDim.y;
      linc2 < conf_rng.volume/ac1;
      linc2 += gridDim.y*blockDim.y)
  {
    // full linear cell index (not including components) is 
    // idx1 + ac1*idx2 + ac1*ac2*idx3 + ... = idx1 + ac1*linc2.
    // we want to find the start linear index of each contiguous data block, 
    // which corresponds to idx1 = 0. 
    // so linear index of start of contiguous block is ac1*linc2.
    if (linc1 < ac1) {
      gkyl_sub_range_inv_idx(&conf_rng, linc1 + ac1*linc2, idx);
      long start = gkyl_range_idx(&conf_rng, idx);

      struct gkyl_mat out_d = gkyl_nmat_get(xs, linc1 + ac1*linc2);
      double *u_d = (double*) gkyl_array_fetch(u_out, start);
      double *vtsq_d = (double*) gkyl_array_fetch(vtsq_out, start);
    
      prim_lbo_copy_sol(&out_d, nc, vdim, u_d, vtsq_d);
    }
  }
}

void
gkyl_prim_lbo_cross_calc_advance_cu(gkyl_prim_lbo_cross_calc* calc,
  struct gkyl_basis cbasis, const struct gkyl_range conf_rng,
  const struct gkyl_array *greene,
  double self_m, const struct gkyl_array *self_u, const struct gkyl_array *self_vtsq,
  double cross_m, const struct gkyl_array *cross_u, const struct gkyl_array *cross_vtsq, 
  const struct gkyl_array *moms, const struct gkyl_array *boundary_corrections, 
  struct gkyl_array *u_out, struct gkyl_array *vtsq_out)
{
  dim3 dimGrid, dimBlock;
  gkyl_get_array_range_kernel_launch_dims(&dimGrid, &dimBlock, conf_rng);
  int nc = cbasis.num_basis;
  int vdim = calc->prim->pdim - calc->prim->cdim;
  int N = nc*(vdim + 1);
  
  if (calc->is_first) {
    calc->As = gkyl_nmat_cu_dev_new(conf_rng.volume, N, N);
    calc->xs = gkyl_nmat_cu_dev_new(conf_rng.volume, N, 1);
    calc->mem = gkyl_nmat_linsolve_lu_cu_dev_new(calc->As->num, calc->As->nr);
    calc->is_first = false;
  }

  gkyl_prim_lbo_cross_calc_set_cu_ker<<<dimGrid, dimBlock>>>(calc->on_dev,
    calc->As->on_dev, calc->xs->on_dev, 
    cbasis, conf_rng, 
    greene->on_dev, 
    self_m, self_u->on_dev, self_vtsq->on_dev,
    cross_m, cross_u->on_dev, cross_vtsq->on_dev,
    moms->on_dev, boundary_corrections->on_dev);
  
  bool status = gkyl_nmat_linsolve_lu_pa(calc->mem, calc->As, calc->xs);
  
  gkyl_prim_lbo_copy_sol_cu_ker<<<dimGrid, dimBlock>>>(calc->xs->on_dev,
    cbasis, conf_rng, nc, vdim, 
    u_out->on_dev, vtsq_out->on_dev);
}

gkyl_prim_lbo_cross_calc*
gkyl_prim_lbo_cross_calc_cu_dev_new(const struct gkyl_rect_grid *grid,
  struct gkyl_prim_lbo_type *prim)
{
  gkyl_prim_lbo_cross_calc *up = (gkyl_prim_lbo_cross_calc*) gkyl_malloc(sizeof(gkyl_prim_lbo_cross_calc));
  up->grid = *grid;
  up->prim = prim;

  up->is_first = true;
  up->As = up->xs = 0;
  up->mem = 0;

  struct gkyl_prim_lbo_type *pt = gkyl_prim_lbo_type_acquire(prim);
  up->prim = pt->on_dev; // so memcpy below gets dev copy

  up->flags = 0;
  GKYL_SET_CU_ALLOC(up->flags);

  gkyl_prim_lbo_cross_calc *up_cu = (gkyl_prim_lbo_cross_calc*) gkyl_cu_malloc(sizeof(gkyl_prim_lbo_cross_calc));
  gkyl_cu_memcpy(up_cu, up, sizeof(gkyl_prim_lbo_cross_calc), GKYL_CU_MEMCPY_H2D);

  up->prim = pt; // host portion of struct should have host copy
  up->on_dev = up_cu; // host pointer
  
  return up;
}
