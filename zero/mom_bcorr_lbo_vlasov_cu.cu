#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <assert.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_util.h>
#include <gkyl_mom_bcorr_lbo_vlasov.h>
#include <gkyl_mom_bcorr_lbo_vlasov_priv.h>
}

enum { f, vf, BAD };

static int
get_mom_id(const char *mom)
{
  int mom_idx = BAD;

  if (strcmp(mom, "f") == 0) { // density
    mom_idx = f;
  }
  else if (strcmp(mom, "vf") == 0) { // momentum
    mom_idx = vf;
  }
  else {
    mom_idx = BAD;
  }

  return mom_idx;
}

__global__
static void
gkyl_mom_bcorr_lbo_vlasov_set_cu_dev_ptrs(struct mom_type_bcorr_lbo_vlasov* mom_bcorr, int mom_id, enum gkyl_basis_type b_type, int vdim, int poly_order, int tblidx)
{
  mom_bcorr->momt.kernel = kernel;

  // choose kernel tables based on basis-function type
  const gkyl_mom_bcorr_lbo_vlasov_kern_list *mom_bcorr_lbo_vlasov_f_kernels, *mom_bcorr_lbo_vlasov_vf_kernels;

  switch (b_type) {
  case GKYL_BASIS_MODAL_SERENDIPITY:
    mom_bcorr_lbo_vlasov_f_kernels = ser_mom_bcorr_lbo_vlasov_f_kernels;
    mom_bcorr_lbo_vlasov_vf_kernels = ser_mom_bcorr_lbo_vlasov_vf_kernels;
    break;

  case GKYL_BASIS_MODAL_TENSOR:
    break;

  default:
    assert(false);
    break;
  }

  switch (mom_id) {
  case f:
    mom_bcorr->kernel = mom_bcorr_lbo_vlasov_f_kernels[tblidx].kernels[poly_order];
    mom_bcorr->momt.num_mom = vdim;
    break;

  case vf:
    mom_bcorr->kernel = mom_bcorr_lbo_vlasov_vf_kernels[tblidx].kernels[poly_order];
    mom_bcorr->momt.num_mom = vdim;
    break;

  default: // can't happen
    break;
  }
}

struct gkyl_mom_type*
gkyl_mom_bcorr_lbo_vlasov_cu_dev_new(const struct gkyl_basis* cbasis,
  const struct gkyl_basis* pbasis, const char *mom, const double *vBoundary)
{
  assert(cbasis->poly_order == pbasis->poly_order);

  struct mom_type_bcorr_lbo_vlasov *mom_bcorr = (struct mom_type_bcorr_lbo_vlasov*) gkyl_malloc(sizeof(struct mom_type_bcorr_lbo_vlasov));

  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  mom_bcorr->momt.cdim = cdim;
  mom_bcorr->momt.pdim = pdim;
  mom_bcorr->momt.poly_order = poly_order;
  mom_bcorr->momt.num_config = cbasis->num_basis;
  mom_bcorr->momt.num_phase = pbasis->num_basis;
  for (int d=0; d<vdim; ++d) {
    mom_bcorr->vBoundary[d] = vBoundary[d];
    mom_bcorr->vBoundary[d + vdim] = vBoundary[d + vdim];
  }

  int mom_id = get_mom_id(mom);
  assert(mom_id != BAD);
  mom_bcorr->momt.num_mom = vdim; // number of moments

  mom_bcorr->momt.flag = 0;
  GKYL_SET_CU_ALLOC(mom_bcorr->momt.flag);
  mom_bcorr->momt.ref_count = gkyl_ref_count_init(gkyl_mom_free);

  // copy struct to device
  struct mom_type_bcorr_lbo_vlasov *mom_bcorr_cu = (struct mom_type_bcorr_lbo_vlasov*)
    gkyl_cu_malloc(sizeof(struct mom_type_bcorr_lbo_vlasov));
  gkyl_cu_memcpy(mom_bcorr_cu, mom_bcorr, sizeof(struct mom_type_bcorr_lbo_vlasov), GKYL_CU_MEMCPY_H2D);

  assert(cv_index[cdim].vdim[vdim] != -1);


  gkyl_mom_bcorr_lbo_vlasov_set_cu_dev_ptrs<<<1,1>>>(mom_bcorr_cu, mom_id, cbasis->b_type,
    vdim, poly_order, cv_index[cdim].vdim[vdim]);

  mom_bcorr->momt.on_dev = &mom_bcorr_cu->momt;

  return &mom_bcorr->momt;
}
