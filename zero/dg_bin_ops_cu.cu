#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_bin_ops.h>
#include <gkyl_dg_bin_ops_priv.h>
#include <gkyl_mat.h>
#include <gkyl_util.h>
}

// start ID for use in various loops
#define START_ID (threadIdx.x + blockIdx.x*blockDim.x)

static void
gkyl_get_array_range_kernel_launch_dims(dim3* dimGrid, dim3* dimBlock, gkyl_range range, int ncomp)
{
  int volume = range.volume;
  int ndim = range.ndim;
  // ac1 = size of last dimension of range (fastest moving dimension)
  int ac1 = range.iac[ndim-1] > 0 ? range.iac[ndim-1] : 1;
  dimBlock->x = min(ncomp*ac1, GKYL_DEFAULT_NUM_THREADS);
  dimGrid->x = gkyl_int_div_up(ncomp*ac1, dimBlock->x);

  dimBlock->y = gkyl_int_div_up(GKYL_DEFAULT_NUM_THREADS, ncomp*ac1);
  dimGrid->y = gkyl_int_div_up(volume, ac1*dimBlock->y);
}

__global__ void
gkyl_dg_mul_op_cu_kernel(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;
  mul_op_t mul_op = choose_ser_mul_kern(ndim, poly_order);

  for (unsigned long linc = START_ID; linc < NSIZE(out); linc += blockDim.x*gridDim.x) {
    
    const double *lop_d = (const double*) gkyl_array_cfetch(lop, linc);
    const double *rop_d = (const double*) gkyl_array_cfetch(rop, linc);
    double *out_d = (double*) gkyl_array_fetch(out, linc);

    mul_op(lop_d+c_lop*num_basis, rop_d+c_rop*num_basis, out_d+c_oop*num_basis);
  }  
}

// Host-side wrapper for dg multiplication operation
void
gkyl_dg_mul_op_cu(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop)
{
  gkyl_dg_mul_op_cu_kernel<<<out->nblocks, out->nthreads>>>(basis, c_oop, out->on_dev, c_lop, lop->on_dev, c_rop, rop->on_dev);
}

__global__ void
gkyl_dg_mul_op_range_cu_kernel(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop, struct gkyl_range range)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;
  mul_op_t mul_op = choose_ser_mul_kern(ndim, poly_order);

  long n = NCOM(out);
  int idx[GKYL_MAX_DIM];
  // ac1 = size of last dimension of range (fastest moving dimension)
  long ac1 = range.iac[ndim-1] > 0 ? range.iac[ndim-1] : 1;

  // 2D thread grid
  // linc1 = c + n*idx1 (contiguous data, including component index c, with idx1 = 0,.., ac1-1)
  long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
  // linc2 = idx2 + ac2*idx3 + ...
  for (unsigned long linc2 = threadIdx.y + blockIdx.y*blockDim.y;
      linc2 < range.volume/ac1;
      linc2 += gridDim.y*blockDim.y)
  {
    // full linear cell index (not including components) is 
    // idx1 + ac1*idx2 + ac1*ac2*idx3 + ... = idx1 + ac1*linc2.
    // we want to find the start linear index of each contiguous data block, 
    // which corresponds to idx1 = 0. 
    // so linear index of start of contiguous block is ac1*linc2.
    gkyl_sub_range_inv_idx(&range, ac1*linc2, idx);
    long start = gkyl_range_idx(&range, idx);

    const double *lop_d = (const double*) gkyl_array_cfetch(lop, start);
    const double *rop_d = (const double*) gkyl_array_cfetch(rop, start);
    double *out_d = (double*) gkyl_array_fetch(out, start);
    // do operation on contiguous data block
    if (linc1 < n*ac1)
      mul_op(lop_d+c_lop*num_basis, rop_d+c_rop*num_basis, out_d+c_oop*num_basis);  
  }
}

// Host-side wrapper for range-based dg multiplication operation
void
gkyl_dg_mul_op_range_cu(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop, struct gkyl_range range)
{
  dim3 dimGrid, dimBlock;
  gkyl_get_array_range_kernel_launch_dims(&dimGrid, &dimBlock, range, out->ncomp);

  gkyl_dg_mul_op_range_cu_kernel<<<dimGrid, dimBlock>>>(basis, c_oop, out->on_dev, c_lop, lop->on_dev, c_rop, rop->on_dev, range);
}

__global__ void
gkyl_dg_div_set_op_cu_kernel(struct gkyl_nmat *As, struct gkyl_nmat *xs,
  struct gkyl_basis basis, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;
  div_set_op_t div_set_op = choose_ser_div_set_kern(ndim, poly_order);

  for (unsigned long linc = START_ID; linc < NSIZE(out); linc += blockDim.x*gridDim.x) {
    
    const double *lop_d = (const double*) gkyl_array_cfetch(lop, linc);
    const double *rop_d = (const double*) gkyl_array_cfetch(rop, linc);

    struct gkyl_mat A = gkyl_nmat_get(As, linc);
    struct gkyl_mat x = gkyl_nmat_get(xs, linc);
    gkyl_mat_clear(&A, 0.0); gkyl_mat_clear(&x, 0.0);
    div_set_op(&A, &x, lop_d+c_lop*num_basis, rop_d+c_rop*num_basis);
  }
}

__global__ void
gkyl_dg_div_copy_sol_op_cu_kernel(struct gkyl_nmat *xs,
  struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out)
{
  int num_basis = basis.num_basis;
  for (unsigned long linc = START_ID; linc < NSIZE(out); linc += blockDim.x*gridDim.x) {
    double *out_d = (double*) gkyl_array_fetch(out, linc);
    struct gkyl_mat x = gkyl_nmat_get(xs, linc);
    binop_div_copy_sol(&x, out_d+c_oop*num_basis);
  }  
}

// Host-side wrapper for dg division operation
void
gkyl_dg_div_op_cu(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop)
{
  int num_basis = basis.num_basis;  
  // allocate memory for use in kernels
  struct gkyl_nmat *A_d = gkyl_nmat_cu_dev_new(out->size, num_basis, num_basis);
  struct gkyl_nmat *x_d = gkyl_nmat_cu_dev_new(out->size, num_basis, 1);

  // construct matrices using CUDA kernel
  gkyl_dg_div_set_op_cu_kernel<<<out->nblocks, out->nthreads>>>(A_d->on_dev, x_d->on_dev,
    basis, out->on_dev, c_lop, lop->on_dev, c_rop, rop->on_dev);
  // invert all matrices in batch mode
  bool status = gkyl_nmat_linsolve_lu(A_d, x_d);
  // copy solution into array (also lives on the device)
  gkyl_dg_div_copy_sol_op_cu_kernel<<<out->nblocks, out->nthreads>>>(x_d->on_dev, basis, c_oop, out->on_dev);

  gkyl_nmat_release(A_d);
  gkyl_nmat_release(x_d);  
}

__global__ void
gkyl_dg_div_set_op_range_cu_kernel(struct gkyl_nmat *As, struct gkyl_nmat *xs,
  struct gkyl_basis basis, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop, struct gkyl_range range)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;
  div_set_op_t div_set_op = choose_ser_div_set_kern(ndim, poly_order);

  long n = NCOM(out);
  int idx[GKYL_MAX_DIM];
  // ac1 = size of last dimension of range (fastest moving dimension)
  long ac1 = range.iac[ndim-1] > 0 ? range.iac[ndim-1] : 1;

  long count = 0;
  // 2D thread grid
  // linc1 = c + n*idx1 (contiguous data, including component index c, with idx1 = 0,.., ac1-1)
  long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
  // linc2 = idx2 + ac2*idx3 + ...
  for (unsigned long linc2 = threadIdx.y + blockIdx.y*blockDim.y;
      linc2 < range.volume/ac1;
      linc2 += gridDim.y*blockDim.y)
  {
    // full linear cell index (not including components) is 
    // idx1 + ac1*idx2 + ac1*ac2*idx3 + ... = idx1 + ac1*linc2.
    // we want to find the start linear index of each contiguous data block, 
    // which corresponds to idx1 = 0. 
    // so linear index of start of contiguous block is ac1*linc2.
    gkyl_sub_range_inv_idx(&range, ac1*linc2, idx);
    long start = gkyl_range_idx(&range, idx);
    
    const double *lop_d = (const double*) gkyl_array_cfetch(lop, start);
    const double *rop_d = (const double*) gkyl_array_cfetch(rop, start);

    struct gkyl_mat A = gkyl_nmat_get(As, count);
    struct gkyl_mat x = gkyl_nmat_get(xs, count);
    gkyl_mat_clear(&A, 0.0); gkyl_mat_clear(&x, 0.0);  
    // do operation on contiguous data block
    if (linc1 < n*ac1)
      div_set_op(&A, &x, lop_d+c_lop*num_basis, rop_d+c_rop*num_basis);

    count += 1;
  }  
}

__global__ void
gkyl_dg_div_copy_sol_op_range_cu_kernel(struct gkyl_nmat *xs,
  struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out, struct gkyl_range range)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  long n = NCOM(out);
  int idx[GKYL_MAX_DIM];

  // ac1 = size of last dimension of range (fastest moving dimension)
  long ac1 = range.iac[ndim-1] > 0 ? range.iac[ndim-1] : 1;

  long count = 0;
  // 2D thread grid
  // linc1 = c + n*idx1 (contiguous data, including component index c, with idx1 = 0,.., ac1-1)
  long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
  // linc2 = idx2 + ac2*idx3 + ...
  for (unsigned long linc2 = threadIdx.y + blockIdx.y*blockDim.y;
      linc2 < range.volume/ac1;
      linc2 += gridDim.y*blockDim.y)
  {
    // full linear cell index (not including components) is 
    // idx1 + ac1*idx2 + ac1*ac2*idx3 + ... = idx1 + ac1*linc2.
    // we want to find the start linear index of each contiguous data block, 
    // which corresponds to idx1 = 0. 
    // so linear index of start of contiguous block is ac1*linc2.
    gkyl_sub_range_inv_idx(&range, ac1*linc2, idx);
    long start = gkyl_range_idx(&range, idx);

    double *out_d = (double*) gkyl_array_fetch(out, start);
    struct gkyl_mat x = gkyl_nmat_get(xs, count);
    // do operation on contiguous data block
    if (linc1 < n*ac1)
      binop_div_copy_sol(&x, out_d+c_oop*num_basis);

    count += 1;
  }  
}

// Host-side wrapper for range-based dg division operation
void
gkyl_dg_div_op_range_cu(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop, struct gkyl_range range)
{
  dim3 dimGrid, dimBlock;
  gkyl_get_array_range_kernel_launch_dims(&dimGrid, &dimBlock, range, out->ncomp);

  int num_basis = basis.num_basis;    
  // allocate memory for use in kernels
  struct gkyl_nmat *A_d = gkyl_nmat_cu_dev_new(range.volume, num_basis, num_basis);
  struct gkyl_nmat *x_d = gkyl_nmat_cu_dev_new(range.volume, num_basis, 1);

  // construct matrices using CUDA kernel  
  gkyl_dg_div_set_op_range_cu_kernel<<<dimGrid, dimBlock>>>(A_d->on_dev, x_d->on_dev,
    basis, out->on_dev, c_lop, lop->on_dev, c_rop, rop->on_dev, range);
  // invert all matrices in batch mode
  bool status = gkyl_nmat_linsolve_lu(A_d, x_d);
  // copy solution into array (also lives on the device)
  gkyl_dg_div_copy_sol_op_range_cu_kernel<<<dimGrid, dimBlock>>>(x_d->on_dev, basis, c_oop, out->on_dev, range);

  gkyl_nmat_release(A_d);
  gkyl_nmat_release(x_d);  
}
