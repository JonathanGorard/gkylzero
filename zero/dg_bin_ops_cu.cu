#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <math.h>
#include <time.h>

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_array_ops_priv.h>
#include <gkyl_dg_bin_ops.h>
#include <gkyl_dg_bin_ops_priv.h>
#include <gkyl_mat.h>
#include <gkyl_util.h>
}

// start ID for use in various loops
#define START_ID (threadIdx.x + blockIdx.x*blockDim.x)

__global__ void
gkyl_dg_mul_op_cu_kernel(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;
  mul_op_t mul_op = choose_ser_mul_kern(ndim, poly_order);

  for (unsigned long linc = START_ID; linc < NSIZE(out); linc += blockDim.x*gridDim.x) {
    
    const double *lop_d = (const double*) gkyl_array_cfetch(lop, linc);
    const double *rop_d = (const double*) gkyl_array_cfetch(rop, linc);
    double *out_d = (double*) gkyl_array_fetch(out, linc);

    mul_op(lop_d+c_lop*num_basis, rop_d+c_rop*num_basis, out_d+c_oop*num_basis);
  }  
}

// Host-side wrapper for dg multiplication operation
void
gkyl_dg_mul_op_cu(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop)
{
  gkyl_dg_mul_op_cu_kernel<<<out->nblocks, out->nthreads>>>(basis, c_oop, out->on_dev,
    c_lop, lop->on_dev, c_rop, rop->on_dev);
}

__global__ void
gkyl_dg_mul_op_range_cu_kernel(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop, struct gkyl_range range)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;
  mul_op_t mul_op = choose_ser_mul_kern(ndim, poly_order);

  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long start = gkyl_range_idx(&range, idx);

    const double *lop_d = (const double*) gkyl_array_cfetch(lop, start);
    const double *rop_d = (const double*) gkyl_array_cfetch(rop, start);
    double *out_d = (double*) gkyl_array_fetch(out, start);

    mul_op(lop_d+c_lop*num_basis, rop_d+c_rop*num_basis, out_d+c_oop*num_basis);
  }
}

// Host-side wrapper for range-based dg multiplication operation
void
gkyl_dg_mul_op_range_cu(struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop, struct gkyl_range *range)
{
  int nblocks = range->nblocks;
  int nthreads = range->nthreads;
  gkyl_dg_mul_op_range_cu_kernel<<<nblocks, nthreads>>>(basis, c_oop, out->on_dev,
    c_lop, lop->on_dev, c_rop, rop->on_dev, *range);
}

__global__ void
gkyl_dg_mul_conf_phase_op_range_cu_kernel(struct gkyl_basis cbasis,
  struct gkyl_basis pbasis, struct gkyl_array* pout,
  const struct gkyl_array* cop, const struct gkyl_array* pop,
  struct gkyl_range crange, struct gkyl_range prange)
{
  int cdim = cbasis.ndim;
  int vdim = pbasis.ndim - cdim;
  int poly_order = cbasis.poly_order;
  mul_op_t mul_op = choose_mul_conf_phase_kern(pbasis.b_type, cdim, vdim, poly_order);

  int pidx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < prange.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&prange, linc1, pidx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long start = gkyl_range_idx(&prange, pidx);

    const double *pop_d = (const double*) gkyl_array_cfetch(pop, start);
    double *pout_d = (double*) gkyl_array_fetch(pout, start);

    int cidx[3];
    for (int d=0; d<cdim; d++) cidx[d] = pidx[d];
    long cstart = gkyl_range_idx(&crange, cidx);
    const double *cop_d = (const double*) gkyl_array_cfetch(cop, cstart);

    mul_op(cop_d, pop_d, pout_d);
  }
}

// Host-side wrapper for range-based dg conf*phase multiplication.
void
gkyl_dg_mul_conf_phase_op_range_cu(struct gkyl_basis *cbasis,
  struct gkyl_basis *pbasis, struct gkyl_array* pout,
  const struct gkyl_array* cop, const struct gkyl_array* pop,
  struct gkyl_range *crange, struct gkyl_range *prange)
{
  int nblocks = prange->nblocks;
  int nthreads = prange->nthreads;
  gkyl_dg_mul_conf_phase_op_range_cu_kernel<<<nblocks, nthreads>>>(*cbasis, *pbasis,
    pout->on_dev, cop->on_dev, pop->on_dev, *crange, *prange);
}

__global__ void
gkyl_dg_div_set_op_cu_kernel(struct gkyl_nmat *As, struct gkyl_nmat *xs,
  struct gkyl_basis basis, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;
  div_set_op_t div_set_op = choose_ser_div_set_kern(ndim, poly_order);

  for (unsigned long linc = START_ID; linc < NSIZE(out); linc += blockDim.x*gridDim.x) {
    
    const double *lop_d = (const double*) gkyl_array_cfetch(lop, linc);
    const double *rop_d = (const double*) gkyl_array_cfetch(rop, linc);

    struct gkyl_mat A = gkyl_nmat_get(As, linc);
    struct gkyl_mat x = gkyl_nmat_get(xs, linc);
    gkyl_mat_clear(&A, 0.0); gkyl_mat_clear(&x, 0.0);
    div_set_op(&A, &x, lop_d+c_lop*num_basis, rop_d+c_rop*num_basis);
  }
}

__global__ void
gkyl_dg_div_copy_sol_op_cu_kernel(struct gkyl_nmat *xs,
  struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out)
{
  int num_basis = basis.num_basis;
  for (unsigned long linc = START_ID; linc < NSIZE(out); linc += blockDim.x*gridDim.x) {
    double *out_d = (double*) gkyl_array_fetch(out, linc);
    struct gkyl_mat x = gkyl_nmat_get(xs, linc);
    binop_div_copy_sol(&x, out_d+c_oop*num_basis);
  }  
}

// Host-side wrapper for dg division operation
void
gkyl_dg_div_op_cu(gkyl_dg_bin_op_mem *mem, struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop)
{
  int num_basis = basis.num_basis;  
  // allocate memory for use in kernels
  struct gkyl_nmat *A_d = mem->As;
  struct gkyl_nmat *x_d = mem->xs;

  // construct matrices using CUDA kernel
  gkyl_dg_div_set_op_cu_kernel<<<out->nblocks, out->nthreads>>>(A_d->on_dev, x_d->on_dev,
    basis, out->on_dev, c_lop, lop->on_dev, c_rop, rop->on_dev);
  // invert all matrices in batch mode
  bool status = gkyl_nmat_linsolve_lu_pa(mem->lu_mem, A_d, x_d);
  // copy solution into array (also lives on the device)
  gkyl_dg_div_copy_sol_op_cu_kernel<<<out->nblocks, out->nthreads>>>(x_d->on_dev, basis, c_oop, out->on_dev);

}

__global__ void
gkyl_dg_div_set_op_range_cu_kernel(struct gkyl_nmat *As, struct gkyl_nmat *xs,
  struct gkyl_basis basis, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop, struct gkyl_range range)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;
  div_set_op_t div_set_op = choose_ser_div_set_kern(ndim, poly_order);

  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long start = gkyl_range_idx(&range, idx);

    const double *lop_d = (const double*) gkyl_array_cfetch(lop, start);
    const double *rop_d = (const double*) gkyl_array_cfetch(rop, start);

    struct gkyl_mat A = gkyl_nmat_get(As, linc1);
    struct gkyl_mat x = gkyl_nmat_get(xs, linc1);
    gkyl_mat_clear(&A, 0.0); gkyl_mat_clear(&x, 0.0);  

    div_set_op(&A, &x, lop_d+c_lop*num_basis, rop_d+c_rop*num_basis);
  }
}

__global__ void
gkyl_dg_div_copy_sol_op_range_cu_kernel(struct gkyl_nmat *xs,
  struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out, struct gkyl_range range)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;

  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long start = gkyl_range_idx(&range, idx);

    double *out_d = (double*) gkyl_array_fetch(out, start);

    struct gkyl_mat x = gkyl_nmat_get(xs, linc1);

    binop_div_copy_sol(&x, out_d+c_oop*num_basis);
  }  
}

// Host-side wrapper for range-based dg division operation
void
gkyl_dg_div_op_range_cu(gkyl_dg_bin_op_mem *mem, struct gkyl_basis basis,
  int c_oop, struct gkyl_array* out,
  int c_lop, const struct gkyl_array* lop,
  int c_rop, const struct gkyl_array* rop, struct gkyl_range range)
{
  int nblocks = range.nblocks;
  int nthreads = range.nthreads;
  int num_basis = basis.num_basis;    
  // allocate memory for use in kernels
  struct gkyl_nmat *A_d = mem->As;
  struct gkyl_nmat *x_d = mem->xs;

  // construct matrices using CUDA kernel  
  gkyl_dg_div_set_op_range_cu_kernel<<<nblocks, nthreads>>>(A_d->on_dev,
    x_d->on_dev, basis, out->on_dev, c_lop, lop->on_dev, c_rop, rop->on_dev, range);
  // invert all matrices in batch mode
  bool status = gkyl_nmat_linsolve_lu_pa(mem->lu_mem, A_d, x_d);
  // copy solution into array (also lives on the device)
  gkyl_dg_div_copy_sol_op_range_cu_kernel<<<nblocks, nthreads>>>(x_d->on_dev,
    basis, c_oop, out->on_dev, range);
}

__global__ void
gkyl_dg_calc_op_range_cu_kernel(struct gkyl_basis basis, int c_oop, struct gkyl_array *out,
  int c_iop, const struct gkyl_array *iop,
  struct gkyl_range range, enum gkyl_dg_op op)
{
  int num_basis = basis.num_basis;
  int ndim = basis.ndim;
  int poly_order = basis.poly_order;

  dp_op_t op_func = dg_get_op_func(op);
  double fact = // factor for rescaling return value of op_func
    op == GKYL_DG_OP_MEAN ? sqrt(pow(2,ndim)) : pow(2,ndim);

  int idx[GKYL_MAX_DIM];

  for (unsigned long linc1 = threadIdx.x + blockIdx.x*blockDim.x;
      linc1 < range.volume;
      linc1 += gridDim.x*blockDim.x)
  {
    // inverse index from linc1 to idx
    // must use gkyl_sub_range_inv_idx so that linc1=0 maps to idx={1,1,...}
    // since update_range is a subrange
    gkyl_sub_range_inv_idx(&range, linc1, idx);

    // convert back to a linear index on the super-range (with ghost cells)
    // linc will have jumps in it to jump over ghost cells
    long start = gkyl_range_idx(&range, idx);

    const double *iop_d = (const double*) gkyl_array_cfetch(iop, start);
    double *out_d = (double*) gkyl_array_fetch(out, start);

    out_d[c_oop] =
      op_func(num_basis, iop_d+c_iop*num_basis)/fact;
  }
}

void
gkyl_dg_calc_op_range_cu(struct gkyl_basis basis, int c_oop, struct gkyl_array *out,
  int c_iop, const struct gkyl_array *iop,
  struct gkyl_range range, enum gkyl_dg_op op)
{
  gkyl_dg_calc_op_range_cu_kernel<<<out->nblocks, out->nthreads>>>(basis, c_oop, out->on_dev,
    c_iop, iop->on_dev, range, op);
}
