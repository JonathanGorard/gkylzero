#include "hip/hip_runtime.h"
/* -*- c++ -*- */

extern "C" {
#include <gkyl_alloc.h>
#include <gkyl_alloc_flags_priv.h>
#include <gkyl_dg_lbo_gyrokinetic_diff.h>    
#include <gkyl_dg_lbo_gyrokinetic_diff_priv.h>
}

#include <cassert>

// CUDA kernel to set pointer to nuSum, sum of collisionalities
// This is required because eqn object lives on device,
// and so its members cannot be modified without a full __global__ kernel on device.
__global__ static void
gkyl_lbo_gyrokinetic_diff_set_auxfields_cu_kernel(const struct gkyl_dg_eqn *eqn, 
  const struct gkyl_array *nuSum, const struct gkyl_array *nuPrimMomsSum, const struct gkyl_array *m2self)
{
  struct dg_lbo_gyrokinetic_diff *lbo_gyrokinetic_diff = container_of(eqn, struct dg_lbo_gyrokinetic_diff, eqn);
  lbo_gyrokinetic_diff->auxfields.nuSum = nuSum;
  lbo_gyrokinetic_diff->auxfields.nuPrimMomsSum = nuPrimMomsSum;
  lbo_gyrokinetic_diff->auxfields.m2self = m2self;
}

//// Host-side wrapper for device kernels setting nuSum, nuUSum and nuVtSqSum.
void
gkyl_lbo_gyrokinetic_diff_set_auxfields_cu(const struct gkyl_dg_eqn *eqn, struct gkyl_dg_lbo_gyrokinetic_diff_auxfields auxin)
{
  gkyl_lbo_gyrokinetic_diff_set_auxfields_cu_kernel<<<1,1>>>(eqn, 
    auxin.nuSum->on_dev, auxin.nuPrimMomsSum->on_dev, auxin.m2self->on_dev);
}

// CUDA kernel to set device pointers to range object and gyrokinetic LBO kernel function
// Doing function pointer stuff in here avoids troublesome hipMemcpyFromSymbol
__global__ static void
dg_lbo_gyrokinetic_diff_set_cu_dev_ptrs(struct dg_lbo_gyrokinetic_diff *lbo_gyrokinetic_diff, enum gkyl_basis_type b_type,
  int cv_index, int cdim, int vdim, int poly_order)
{
  lbo_gyrokinetic_diff->auxfields.nuSum = 0; 
  lbo_gyrokinetic_diff->auxfields.nuPrimMomsSum = 0; 
  lbo_gyrokinetic_diff->auxfields.m2self = 0; 

  lbo_gyrokinetic_diff->eqn.surf_term = surf;
  lbo_gyrokinetic_diff->eqn.boundary_surf_term = boundary_surf;

  const gkyl_dg_lbo_gyrokinetic_diff_vol_kern_list *vol_kernels;
  const gkyl_dg_lbo_gyrokinetic_diff_surf_kern_list *surf_vpar_kernels, *surf_mu_kernels;
  const gkyl_dg_lbo_gyrokinetic_diff_boundary_surf_kern_list *boundary_surf_vpar_kernels, *boundary_surf_mu_kernels;

  switch (b_type) {
    case GKYL_BASIS_MODAL_SERENDIPITY:
      vol_kernels = ser_vol_kernels;
      surf_vpar_kernels = ser_surf_vpar_kernels;
      surf_mu_kernels = ser_surf_mu_kernels;
      boundary_surf_vpar_kernels = ser_boundary_surf_vpar_kernels;
      boundary_surf_mu_kernels = ser_boundary_surf_mu_kernels;
      break;

    default:
      assert(false);
      break;    
  }  

  lbo_gyrokinetic_diff->eqn.vol_term = vol_kernels[cv_index].kernels[poly_order];

  lbo_gyrokinetic_diff->surf[0] = surf_vpar_kernels[cv_index].kernels[poly_order];
  if (vdim>1)
    lbo_gyrokinetic_diff->surf[1] = surf_mu_kernels[cv_index].kernels[poly_order];

  lbo_gyrokinetic_diff->boundary_surf[0] = boundary_surf_vpar_kernels[cv_index].kernels[poly_order];
  if (vdim>1)
    lbo_gyrokinetic_diff->boundary_surf[1] = boundary_surf_mu_kernels[cv_index].kernels[poly_order];

}

struct gkyl_dg_eqn*
gkyl_dg_lbo_gyrokinetic_diff_cu_dev_new(const struct gkyl_basis* cbasis, const struct gkyl_basis* pbasis,
  const struct gkyl_range* conf_range, const struct gkyl_rect_grid *pgrid, double mass, const struct gk_geometry *gk_geom)
{
  struct dg_lbo_gyrokinetic_diff *lbo_gyrokinetic_diff =
    (struct dg_lbo_gyrokinetic_diff*) gkyl_malloc(sizeof(struct dg_lbo_gyrokinetic_diff));

  int cdim = cbasis->ndim, pdim = pbasis->ndim, vdim = pdim-cdim;
  int poly_order = cbasis->poly_order;

  lbo_gyrokinetic_diff->cdim = cdim;
  lbo_gyrokinetic_diff->pdim = pdim;

  lbo_gyrokinetic_diff->eqn.num_equations = 1;
  lbo_gyrokinetic_diff->mass = mass;
  // acquire pointer to geometry object
  struct gk_geometry *geom = gkyl_gk_geometry_acquire(gk_geom);
  lbo_gyrokinetic_diff->gk_geom = geom->on_dev; // this is so the memcpy below has geometry on_dev
  lbo_gyrokinetic_diff->conf_range = *conf_range;

  lbo_gyrokinetic_diff->vparMax = pgrid->upper[cdim];
  lbo_gyrokinetic_diff->vparMaxSq = pow(pgrid->upper[cdim],2);
  lbo_gyrokinetic_diff->num_cbasis = cbasis->num_basis;

  lbo_gyrokinetic_diff->eqn.flags = 0;
  GKYL_SET_CU_ALLOC(lbo_gyrokinetic_diff->eqn.flags);
  lbo_gyrokinetic_diff->eqn.ref_count = gkyl_ref_count_init(gkyl_lbo_gyrokinetic_diff_free);

  // copy the host struct to device struct
  struct dg_lbo_gyrokinetic_diff *lbo_gyrokinetic_diff_cu =
    (struct dg_lbo_gyrokinetic_diff*) gkyl_cu_malloc(sizeof(struct dg_lbo_gyrokinetic_diff));

  gkyl_cu_memcpy(lbo_gyrokinetic_diff_cu, lbo_gyrokinetic_diff,
    sizeof(struct dg_lbo_gyrokinetic_diff), GKYL_CU_MEMCPY_H2D);

  dg_lbo_gyrokinetic_diff_set_cu_dev_ptrs<<<1,1>>>(lbo_gyrokinetic_diff_cu,
    cbasis->b_type, cv_index[cdim].vdim[vdim], cdim, vdim, poly_order);

  lbo_gyrokinetic_diff->eqn.on_dev = &lbo_gyrokinetic_diff_cu->eqn;  
  // updater should store host pointers
  lbo_gyrokinetic_diff->gk_geom = geom;
  
  return &lbo_gyrokinetic_diff->eqn;
}
